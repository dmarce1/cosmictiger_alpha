
#include <hpxfft/fourier.hpp>

namespace hpxfft {

void cuda_set_device() {
	int count;
	CUDA_CHECK(hipGetDeviceCount(&count));
	const int device_num = hpxfft::hpx_rank() % count;
	CUDA_CHECK(hipSetDevice(device_num));
}


}
