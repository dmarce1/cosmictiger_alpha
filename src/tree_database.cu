#include "hip/hip_runtime.h"
#define TREE_DATABASE_CU
#include <cosmictiger/tree.hpp>
#include <cosmictiger/tree_database.hpp>
#include <cosmictiger/global.hpp>
#include <cosmictiger/memory.hpp>

#include <cmath>
#include <atomic>

static const int min_trees = 1024 * 1024;

static std::atomic<int> next_chunk;

int hardware_concurrency();

void tree_data_initialize() {
	gpu_tree_data_.chunk_size = 1;
	gpu_tree_data_.ntrees = 10 * global().opts.nparts / GROUP_BUCKET_SIZE / 3;
	gpu_tree_data_.ntrees = std::max(gpu_tree_data_.ntrees, min_trees);
	const int target_chunk_size = gpu_tree_data_.ntrees / (16 * OVERSUBSCRIPTION * hardware_concurrency());
	while (gpu_tree_data_.chunk_size < target_chunk_size) {
		gpu_tree_data_.chunk_size *= 2;
	}
	gpu_tree_data_.nchunks = gpu_tree_data_.ntrees / gpu_tree_data_.chunk_size;

	CUDA_CHECK(hipMemAdvise(&gpu_tree_data_, sizeof(gpu_tree_data_), hipMemAdviseSetReadMostly, 0));

	printf("Allocating %i trees in %i chunks of %i each\n", gpu_tree_data_.ntrees, gpu_tree_data_.nchunks,
			gpu_tree_data_.chunk_size);

	CUDA_MALLOC(gpu_tree_data_.data, gpu_tree_data_.ntrees);
	CUDA_MALLOC(gpu_tree_data_.parts, gpu_tree_data_.ntrees);
	CUDA_MALLOC(gpu_tree_data_.multi, gpu_tree_data_.ntrees);
	CUDA_MALLOC(gpu_tree_data_.ranges, gpu_tree_data_.ntrees);
	CUDA_MALLOC(gpu_tree_data_.active_nodes, gpu_tree_data_.ntrees);
	CUDA_MALLOC(gpu_tree_data_.active_parts, gpu_tree_data_.ntrees);

	tree_data_clear();

	cpu_tree_data_ = gpu_tree_data_;

}

void tree_database_set_readonly() {
#ifdef USE_READMOSTLY
	CUDA_CHECK(hipMemAdvise(gpu_tree_data_.data, gpu_tree_data_.ntrees, hipMemAdviseSetReadMostly, 0));
	CUDA_CHECK(hipMemAdvise(gpu_tree_data_.active_nodes, gpu_tree_data_.ntrees, hipMemAdviseSetReadMostly, 0));
#endif
}

void tree_database_unset_readonly() {
#ifdef USE_READMOSTLY
	CUDA_CHECK(hipMemAdvise(gpu_tree_data_.data, gpu_tree_data_.ntrees, hipMemAdviseUnsetReadMostly, 0));
	CUDA_CHECK(hipMemAdvise(gpu_tree_data_.active_nodes, gpu_tree_data_.ntrees, hipMemAdviseUnsetReadMostly, 0));
#endif
}

void tree_data_clear() {
	next_chunk = 0;
	for (int i = 0; i < gpu_tree_data_.ntrees; i++) {
		gpu_tree_data_.data[i].children[0].dindex = -1;
	}
}


std::pair<int, int> tree_data_allocate() {
	std::pair<int, int> rc;
	const int chunk = next_chunk++;
	if (chunk >= gpu_tree_data_.nchunks) {
		printf("Fatal error - tree arena full!\n");
		abort();
	}
	rc.first = chunk * gpu_tree_data_.chunk_size;
	rc.second = rc.first + gpu_tree_data_.chunk_size;
	return rc;
}

double tree_data_use() {
	return (double) next_chunk / (double) gpu_tree_data_.nchunks;
}

