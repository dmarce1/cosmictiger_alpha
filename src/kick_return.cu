#include "hip/hip_runtime.h"


#include <cosmictiger/kick_return.hpp>
#include <hip/hip_runtime.h>


static __managed__ kick_return gpu_return;


__global__ void kick_return_init_kernel(int min_rung) {
	gpu_return.min_rung = min_rung;
	for( int i = 0; i < MAX_RUNG; i++) {
		gpu_return.rung_cnt[i] = 0;
	}
}


void kick_return_init_gpu(int min_rung) {
	kick_return_init_kernel<<<1,1>>>(min_rung);
	CUDA_CHECK(hipDeviceSynchronize());
}

kick_return kick_return_get_gpu() {
	return gpu_return;
}

__device__ void kick_return_update_rung_gpu(int rung) {
	atomicAdd(&gpu_return.rung_cnt[rung], 1);
}
