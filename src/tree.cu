#include "hip/hip_runtime.h"
struct ewald_indices;
struct periodic_parts;
#include <cosmictiger/cuda.hpp>
#include <stack>

CUDA_DEVICE ewald_indices *four_indices_ptr;
CUDA_DEVICE ewald_indices *real_indices_ptr;
CUDA_DEVICE periodic_parts *periodic_parts_ptr;

#define TREECU
#include <cosmictiger/tree.hpp>
#include <cosmictiger/interactions.hpp>
#include <functional>
#include <cosmictiger/gravity.hpp>

//CUDA_KERNEL cuda_kick()

#define CC_CP_DIRECT 0
#define CC_CP_EWALD 1
#define PC_PP_DIRECT 2
#define PC_PP_EWALD 3
#define N_INTERACTION_TYPES 4

CUDA_DEVICE particle_set *parts;

__managed__ double pp_interaction_time;
__managed__ double pc_interaction_time;
__managed__ double cp_interaction_time;
__managed__ double cc_interaction_time;
__managed__ double total_time;

#define MI 0
#define CI 1
#define OI 2
#define PI 3

CUDA_DEVICE kick_return
cuda_kick(kick_params_type * params_ptr)
{
   kick_params_type &params = *params_ptr;
   __shared__ volatile
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   //  printf( "%i\n", params_ptr->depth);
   //   if( params_ptr->depth > TREE_MAX_DEPTH || params_ptr->depth < 0 ) {
//      printf( "%li\n", params_ptr->depth);
   //  }
   tree_ptr tptr = params.tptr;
   tree& me = *((tree*) tptr);
   const int &tid = threadIdx.x;
   kick_return rc;
   auto &F = params.F;
   auto &L = params.L[params.depth];
   if( tid == 0 ) {
      const auto &Lpos = params.Lpos[params.depth];
      array<float, NDIM> dx;
      for (int dim = 0; dim < NDIM; dim++) {
         const auto x1 = me.pos[dim];
         const auto x2 = Lpos[dim];
         dx[dim] = distance(x1, x2);
      }
      shift_expansion(L,dx);
   }

#ifdef COUNT_FLOPS
   int flops = 0;
#endif
   if (((tree*) tptr)->children[0].ptr == 0) {
      for (int k = tid; k < MAX_BUCKET_SIZE; k += KICK_BLOCK_SIZE) {
         for (int dim = 0; dim < NDIM; dim++) {
            F[dim][k] = 0.f;
         }
      }
      __syncwarp();
   }
   {
      auto &indices = shmem.indices;
      auto &count = shmem.count;

      const auto theta2 = params.theta * params.theta;
      array<vector<tree_ptr>*, NITERS> lists;
      auto &multis = params.multi_interactions;
      auto &parti = params.part_interactions;
      auto &next_checks = params.next_checks;
      auto &opened_checks = params.opened_checks;
      lists[MI] = &multis;
      lists[PI] = &parti;
      lists[CI] = &next_checks;
      lists[OI] = &opened_checks;
      for( int i = 0; i < NITERS; i++) {
         lists[i]->resize(0);
      }
      const auto myradius = SINK_BIAS * ((tree*) tptr)->radius;
      const auto &mypos = ((tree*) tptr)->pos;
      const bool iamleaf = ((tree*) tptr)->children[0].ptr;
      int ninteractions = iamleaf == 0 ? 4 : 2;
      for (int type = 0; type < ninteractions; type++) {
         const bool ewald_dist = type == PC_PP_EWALD || type == CC_CP_EWALD;
         auto& checks = ewald_dist ? params.echecks : params.dchecks;
         const bool direct = type == PC_PP_EWALD || type == PC_PP_DIRECT;
         if (tid < NITERS) {
            count[tid] = 0;
         }
         for( int i = 0; i < NITERS; i++) {
            lists[i]->resize(0);
         }
         __syncwarp();
         int check_count;
         do {
            check_count = checks.size();
            flops += check_count * FLOPS_OPEN;
            if (check_count) {
               const int cimax = ((check_count - 1) / KICK_BLOCK_SIZE + 1) * KICK_BLOCK_SIZE;
               for (int ci = tid; ci < cimax; ci += KICK_BLOCK_SIZE) {
                  for (int i = 0; i < NITERS; i++) {
                     indices[i][tid + 1] = 0;
                  }
                  __syncwarp();
                  if (tid < NITERS) {
                     indices[tid][0] = 0;
                  }
                  __syncwarp();
                  int list_index;
                  const auto h = params.hsoft;
                  if (ci < check_count) {
                     auto &check = checks[ci];
                     const auto &other_radius = ((const tree*) check)->radius;
                     const auto &other_pos = ((const tree*) check)->pos;
                     float d2 = 0.f;
                     const float R2 = sqr(other_radius + myradius + h);                 // 2
#ifdef PERIODIC_OFF
                     for (int dim = 0; dim < NDIM; dim++) {                         // 3
                        d2 += sqr(other_pos[dim].to_float() - mypos[dim].to_float());
                     }
#else
                     for (int dim = 0; dim < NDIM; dim++) {                         // 3
                        d2 += sqr(distance(other_pos[dim], mypos[dim]));
                     }
#endif
                     if (ewald_dist) {
                        d2 = fmaxf(d2, EWALD_MIN_DIST2);                            // 1
                     }
                     const bool far = R2 < theta2 * d2;                             // 2
                     const bool far2 = sqr(other_radius*params.theta + myradius + h) < theta2 * d2;
                     const bool far3 = sqr(other_radius + myradius*params.theta/SINK_BIAS + h) < theta2 * d2;
                     //     const bool isleaf = ((const tree*) check)->children[0].ptr == 0;
                     const bool isleaf = ((const tree*) check)->parts.second - ((const tree*) check)->parts.first <= GROUP_SIZE;
                     auto& other_opened = check.opened;
                     const auto& me_opened = direct;
                     if( !me_opened && far && !other_opened ) {
                        list_index = MI;                                               //CC
                     } else if ( me_opened && far3 && !other_opened ) {
                        list_index = MI;                                               //PC
                     } else if( !me_opened && far2 && isleaf && !iamleaf) {
                        list_index = PI;                                               //CP
                     } else  if(  me_opened && other_opened ) {
                        if( far3 ) {
                           list_index = MI;
                        } else {
                           list_index = PI;
                        }
                     } else if( isleaf ) {
                        other_opened++;
                        list_index = OI;
                     } else {
                        list_index = CI;
                     }
                     indices[list_index][tid + 1] = 1;
                  }
                  __syncwarp();
                  for (int P = 1; P < KICK_BLOCK_SIZE; P *= 2) {
                     array<int, NITERS> tmp;
                     if (tid - P + 1 >= 0) {
                        for (int i = 0; i < NITERS; i++) {
                           tmp[i] = indices[i][tid - P + 1];
                        }
                     }
                     __syncwarp();
                     if (tid - P + 1 >= 0) {
                        for (int i = 0; i < NITERS; i++) {
                           indices[i][tid + 1] += tmp[i];
                        }
                     }
                     __syncwarp();
                  }
                  __syncwarp();
                  for( int i = 0; i < NITERS; i++) {
                     assert(indices[i][tid] <= indices[i][tid+1] );
                     lists[i]->resize(count[i]+indices[i][KICK_BLOCK_SIZE]);
                  }
                  if( ci < check_count ) {
                     const auto &check = checks[ci];
                     assert(count[list_index] + indices[list_index][tid] >= 0);
                     //          printf( "%i %i\n",(*lists[list_index]).size(), count[list_index] + indices[list_index][tid] );
                     (*lists[list_index])[count[list_index] + indices[list_index][tid]] = check;
                  }
                  __syncwarp();
                  if (tid < NITERS) {
                     count[tid] += indices[tid][KICK_BLOCK_SIZE];
                  }
                  __syncwarp();
               }
               __syncwarp();
               check_count = 2 * count[CI];
               checks.resize(check_count);
               for (int i = tid; i < count[CI]; i += KICK_BLOCK_SIZE) {
                  const auto children = next_checks[i].get_children();
                  for (int j = 0; j < NCHILD; j++) {
                     checks[2 * i + j] = children[j];
                  }
               }
               __syncwarp();
               check_count += count[OI];
               checks.resize(check_count);
               for (int i = tid; i < count[OI]; i += KICK_BLOCK_SIZE) {
                  checks[2 * count[CI] + i] = opened_checks[i];
               }
               __syncwarp();
               if (tid == 0) {
                  count[CI] = 0;
                  count[OI] = 0;
               }
               __syncwarp();
               next_checks.resize(0);
               opened_checks.resize(0);
            }
         }while (direct && check_count);
//         if( tid == 0 ) {
//            printf( "%i %i %i\n", params.depth, count[MI], count[PI]);
//         }
         __syncwarp();
         switch (type) {
            case PC_PP_DIRECT:
            //          printf( "%li %li\n", multis.size(), parti.size());
            flops += cuda_pc_interactions(parts,multis, params_ptr);
            flops += cuda_pp_interactions(parts,parti, params_ptr);
            //          if( tid == 0 ) {
            //                          printf( "%i %i %i\n", params_ptr->depth, parti.size(), multis.size());
//            }
            break;
            case CC_CP_DIRECT:
            flops += cuda_cc_interactions(parts,multis, params_ptr);
            flops += cuda_cp_interactions(parts,parti,params_ptr);
            //          if( tid == 0 ) {
            //                        printf( "%i %i %i\n", params_ptr->depth, parti.size(), multis.size());
            //         }
            break;

            case PC_PP_EWALD:
            if(count[PI] > 0 ) {
               //        printf( "PP Ewald should not exist\n");
               //  __trap();
            }
            if(count[MI] > 0 ) {
               //      printf( "PC Ewald should not exist\n");
               //   __trap();
            }
            break;
            case CC_CP_EWALD:
#ifndef PERIODIC_OFF
            if(count[PI] > 0 ) {
               printf( "CP Ewald should not exist\n");
               //     __trap();
            }
            if( count[MI] > 0 ) {
               flops += cuda_ewald_cc_interactions(parts,params_ptr, &shmem.Lreduce);
            }
#endif
            break;
         }
      }
      rc.flops = flops;
   }
   if (!(((tree*) tptr)->children[0].ptr == 0)) {
      params.dchecks.push_top();
      params.echecks.push_top();
      if (tid == 0) {
         params.depth++;
         params.L[params.depth] = L;
         params.Lpos[params.depth] = me.pos;
         params.tptr = ((tree*) tptr)->children[RIGHT];
      }
      __syncwarp();
      kick_return rc1 = cuda_kick(params_ptr);
      if (tid == 0) {
         params.L[params.depth] = L;
         params.tptr = ((tree*) tptr)->children[LEFT];
      }
      __syncwarp();
      params.dchecks.pop_top();
      params.echecks.pop_top();
      kick_return rc2 = cuda_kick(params_ptr);
      if (tid == 0) {
         params.depth--;
      }
      __syncwarp();
      rc.rung = max(rc1.rung, rc2.rung);
      rc.flops += rc1.flops + rc2.flops;
      //   printf( "%li\n", rc.flops);
   } else {
      auto& rungs = shmem.rungs;
      rungs[tid] = 0;
      const auto& myparts = ((tree*)params.tptr)->parts;
      const auto invlog2 = 1.0f / logf(2);
      for (int k = tid; k < myparts.second - myparts.first; k += KICK_BLOCK_SIZE) {
         const auto this_rung = parts->rung(k+myparts.first);
         if( this_rung >= params.rung ) {

            array<float,NDIM> g;
            float phi;
            array<float,NDIM> dx;
            for (int dim = 0; dim < NDIM; dim++) {
               const auto x2 = me.pos[dim];
               const auto x1 = parts->pos(dim,k+myparts.first);
               dx[dim] = distance(x1, x2);
            }
            shift_expansion(L, g, phi, dx);
            for (int dim = 0; dim < NDIM; dim++) {
               F[dim][k] += g[dim];
            }
#ifdef TEST_FORCE
            for( int dim = 0; dim < NDIM; dim++) {
               parts->force(dim,k+myparts.first) = F[dim][k];
            }
#endif
            float dt = params.t0 / (1<<this_rung);
            for( int dim = 0; dim < NDIM; dim++) {
               parts->vel(dim,k+myparts.first) += 0.5 * dt * F[dim][k];
            }
            float fmag = 0.0;
            for( int dim = 0; dim < NDIM; dim++) {
               fmag += sqr(F[dim][k]);
            }
            fmag = sqrtf(fmag);
            //   printf( "%e\n", fmag);
            assert(fmag > 0.0);
            dt = fminf(sqrt(params.scale * params.eta / fmag), params.t0);
            int new_rung = fmaxf(fmaxf(ceil(logf(params.t0/dt) * invlog2), this_rung-1),params.rung);
            dt = params.t0 / (1<<new_rung);
            for( int dim = 0; dim < NDIM; dim++) {
               parts->vel(dim,k+myparts.first) += 0.5 * dt * F[dim][k];
            }
            rungs[tid] = fmaxf(rungs[tid],new_rung);
            parts->set_rung(new_rung, k+myparts.first);
         }
      }
      __syncwarp();
      for( int P = KICK_BLOCK_SIZE/2; P>=1; P/=2) {
         if( tid < P) {
            rungs[tid] = fmaxf(rungs[tid], rungs[tid+P]);
         }
         __syncwarp();
      }
      rc.rung = rungs[0];
   }
   return rc;
}

CUDA_KERNEL cuda_set_kick_params_kernel(particle_set *p, ewald_indices *real_indices, ewald_indices *four_indices,
      periodic_parts *periodic_parts) {
   if (threadIdx.x == 0) {
      parts = p;
      four_indices_ptr = four_indices;
      real_indices_ptr = real_indices;
      periodic_parts_ptr = periodic_parts;
      expansion_init();
      pp_interaction_time = pc_interaction_time = cp_interaction_time = cc_interaction_time = 0.0;

   }
}
void tree::cuda_set_kick_params(particle_set *p, ewald_indices *real_indices, ewald_indices *four_indices,
      periodic_parts *parts) {
cuda_set_kick_params_kernel<<<1,1>>>(p,real_indices, four_indices, parts);
                                                                                          CUDA_CHECK(hipDeviceSynchronize());
}

#ifdef TIMINGS
extern __managed__ double pp_crit1_time;
extern __managed__ double pp_crit2_time;
#endif

CUDA_KERNEL cuda_kick_kernel(kick_return *res, kick_params_type *params) {
   const int &bid = blockIdx.x;
#ifdef TIMINGS
   auto tm = clock64();
#endif
   res[bid] = cuda_kick(params + bid);
   __syncwarp();
   if (threadIdx.x == 0) {
      //     printf( "Kick done\n");
      params[bid].kick_params_type::~kick_params_type();
      //   printf("%e %e %e %e %e\n", walk_time/total_time, pp_interaction_time/total_time, pc_interaction_time/total_time, cp_interaction_time/total_time, cc_interaction_time/total_time);
   }

}

thread_local static std::stack<hipStream_t> streams;

hipStream_t get_stream() {
   if (streams.empty()) {
      hipStream_t stream;
      CUDA_CHECK(hipStreamCreate(&stream));
      streams.push(stream);
   }
   auto stream = streams.top();
   streams.pop();
   return stream;
}

void cleanup_stream(hipStream_t s) {
   streams.push(s);
}

CUDA_KERNEL cuda_ewald_cc_kernel(kick_params_type **params_ptr) {
   __shared__
   volatile
   extern int shmem_ptr[];
   cuda_ewald_shmem &shmem = *((cuda_ewald_shmem*) (shmem_ptr));
   const int &bid = blockIdx.x;
   auto pptr = params_ptr[bid];
   auto rc = cuda_ewald_cc_interactions(parts, pptr, &shmem.Lreduce);
   __syncwarp();
   if (threadIdx.x == 0) {
      params_ptr[bid]->flops = rc;
   }
}

std::function<bool()> cuda_execute_ewald_kernel(kick_params_type **params_ptr, int grid_size) {
   auto stream = get_stream();
   /***/cuda_ewald_cc_kernel<<<grid_size,KICK_BLOCK_SIZE,sizeof(cuda_ewald_shmem),stream>>>(params_ptr);

   struct cuda_ewald_future_shared {
      hipStream_t stream;
      int grid_size;
      mutable bool ready;
   public:
      cuda_ewald_future_shared() {
         ready = false;
      }
      bool operator()() const {
         if (!ready) {
            if (hipStreamQuery(stream) == hipSuccess) {
               ready = true;
               CUDA_CHECK(hipStreamSynchronize(stream));
               cleanup_stream(stream);
            }
         }
         return ready;
      }
   };

   cuda_ewald_future_shared fut;
   fut.stream = stream;
   fut.grid_size = grid_size;
   std::function < bool() > ready_func = [fut]() {
      return fut();
   };
   return ready_func;
}

std::pair<std::function<bool()>, kick_return*> cuda_execute_kick_kernel(kick_params_type *params, int grid_size,
      hipStream_t stream) {
   const size_t shmemsize = sizeof(cuda_kick_shmem);
   unified_allocator alloc;
   kick_return *returns = (kick_return*) alloc.allocate(grid_size * sizeof(kick_return));
// printf( "a\n");
//  CUDA_MALLOC(returns, grid_size);
// printf( "b\n");
//  printf( "Shmem = %li\n", shmemsize);
   /***************************************************************************************************************************************************/
   /**/cuda_kick_kernel<<<grid_size, KICK_BLOCK_SIZE, shmemsize, stream>>>(returns,params);/**/
   /***************************************************************************************************************************************************/
// printf( "c\n");
   struct cuda_kick_future_shared {
      hipStream_t stream;
      kick_return *returns;
      int grid_size;
      mutable bool ready;
   public:
      cuda_kick_future_shared() {
         ready = false;
      }
      bool operator()() const {
         if (!ready) {
            if (hipStreamQuery(stream) == hipSuccess) {
               ready = true;
               CUDA_CHECK(hipStreamSynchronize(stream));
               cleanup_stream(stream);
            }
         }
         return ready;
      }
   };
// printf( "d\n");

   cuda_kick_future_shared fut;
   fut.returns = returns;
   fut.stream = stream;
   fut.grid_size = grid_size;
   std::function < bool() > ready_func = [fut]() {
      return fut();
   };
// printf( "e\n");

   return std::make_pair(std::move(ready_func), std::move(fut.returns));
}

