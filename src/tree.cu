#include "hip/hip_runtime.h"
#include <cosmictiger/tree.hpp>
#include <cosmictiger/cuda.hpp>
#include <functional>

//CUDA_KERNEL cuda_kick()

#define CC_CP_DIRECT 0
#define CC_CP_EWALD 1
#define PC_PP_DIRECT 2
#define PC_PP_EWALD 3
#define N_INTERACTION_TYPES 4

CUDA_DEVICE float theta;
CUDA_DEVICE int8_t rung;

CUDA_DEVICE
void reduce_indexes(array<int, KICK_BLOCK_SIZE + 1> &counts) {
   const int &tid = threadIdx.x;
   counts[0] = 0;
   CUDA_SYNC();
   for (int P = 1; P < KICK_BLOCK_SIZE; P *= 2) {
      int tmp;
      if (tid - P + 1 >= 0) {
         tmp = counts[tid - P + 1];
      }
      CUDA_SYNC();
      if (tid - P + 1 >= 0) {
         counts[tid + 1] += tmp;
      }
      CUDA_SYNC();
   }
}

CUDA_DEVICE kick_return cuda_kick(tree_ptr tptr, kick_stack &stacks, kick_workspace_t &workspace, int depth) {
   const int &tid = threadIdx.x;
   kick_return rc;
   __shared__ array<int, KICK_BLOCK_SIZE + 1>
   mindices;
   __shared__ array<int, KICK_BLOCK_SIZE + 1>
   cindices;
   __shared__ array<int, KICK_BLOCK_SIZE + 1>
   pindices;
   const auto theta2 = theta * theta;
   array<checks_type*, N_INTERACTION_TYPES> all_checks;
   all_checks[CC_CP_DIRECT] = &stacks.dchecks[depth];
   all_checks[CC_CP_EWALD] = &stacks.echecks[depth];
   all_checks[PC_PP_DIRECT] = &stacks.dchecks[depth];
   all_checks[PC_PP_EWALD] = &stacks.echecks[depth];
   auto &multis = workspace.multi_interactions;
   auto &parti = workspace.part_interactions;
   auto &next_checks = workspace.next_checks;
   int ninteractions = ((tree*) tptr)->children[0] == tree_ptr() ? 4 : 2;
   for (int type = 0; type < ninteractions; type++) {
      auto &checks = *(all_checks[type]);
      if (tid == 0) {
         next_checks.resize(0);
         multis.resize(0);
         parti.resize(0);
      }
      CUDA_SYNC();
      const bool ewald_dist = type == PC_PP_EWALD || type == CC_CP_EWALD;
      const bool direct = type == PC_PP_EWALD || type == PC_PP_DIRECT;
      do {
       const int cimax = ((checks.size() - 1) / KICK_BLOCK_SIZE + 1) * KICK_BLOCK_SIZE;
         for (int ci = tid; ci < cimax; ci += KICK_BLOCK_SIZE) {
            mindices[tid + 1] = cindices[tid + 1] = pindices[tid + 1] = 0;
            if (ci < checks.size()) {
               const auto other_radius = ((tree*) checks[ci])->radius;
               const auto other_pos = ((tree*) checks[ci])->pos;
               float d2 = 0.f;
               const float R2 = sqr(other_radius + ((tree*) tptr)->radius);
               for (int dim = 0; dim < NDIM; dim++) {
                  d2 += sqr(fixed<int32_t>(other_pos[dim]) - fixed<int32_t>(((tree*) tptr)->pos[dim])).to_float();
               }
               if (ewald_dist) {
                  d2 = fmaxf(d2, EWALD_MIN_DIST2);
               }
               const bool far = R2 < theta2 * d2;
               if (far) {
                  mindices[tid + 1] = 1;
               } else if (!checks[ci].is_leaf()) {
                  cindices[tid + 1] = 1;
               } else {
                  pindices[tid + 1] = 1;
               }
            }
            reduce_indexes (mindices);
            reduce_indexes (cindices);
            reduce_indexes (pindices);
            int moffset = multis.size();
            int poffset = parti.size();
            int coffset = next_checks.size();
            CUDA_SYNC();
            if (tid == 0) {
               multis.resize(multis.size() + mindices[KICK_BLOCK_SIZE]);
               parti.resize(parti.size() + pindices[KICK_BLOCK_SIZE]);
               next_checks.resize(next_checks.size() + 2 * cindices[KICK_BLOCK_SIZE]);
            }
            CUDA_SYNC();
            if (ci < checks.size()) {
               if (mindices[tid] != mindices[tid + 1]) {
                  assert(mindices[tid] < mindices[tid + 1]);
                  multis[moffset + mindices[tid]] = checks[ci];
               } else if (pindices[tid] != pindices[tid + 1]) {
                  assert(pindices[tid] < pindices[tid + 1]);
                  parti[poffset + pindices[tid]] = checks[ci];
               } else if (cindices[tid] != cindices[tid + 1]) {
                  assert(cindices[tid] < cindices[tid + 1]);
                  next_checks[coffset + 2 * cindices[tid] + LEFT] = checks[ci].get_children()[LEFT];
                  next_checks[coffset + 2 * cindices[tid] + RIGHT] = checks[ci].get_children()[RIGHT];
               }
            }
            CUDA_SYNC();
         }
         CUDA_SYNC();
         if (tid == 0) {
            checks = std::move(next_checks);
         }
         CUDA_SYNC();
      } while (direct && checks.size());

      /*********** DO INTERACTIONS *********************/

   }
   if (!(((tree*) tptr)->children[0] == tree_ptr())) {
      stacks.dchecks[depth + 1] = stacks.dchecks[depth];
      stacks.echecks[depth + 1] = stacks.echecks[depth];
      kick_return rc1 = cuda_kick(((tree*) tptr)->children[LEFT], stacks, workspace, depth + 1);
      stacks.dchecks[depth + 1] = std::move(stacks.dchecks[depth]);
      stacks.echecks[depth + 1] = std::move(stacks.echecks[depth]);
      kick_return rc2 = cuda_kick(((tree*) tptr)->children[RIGHT], stacks, workspace, depth + 1);
      rc.rung = max(rc1.rung, rc2.rung);
   } else {
      rc.rung = 0;
   }
   return rc;
}

CUDA_KERNEL cuda_set_kick_params(float theta_, int rung_) {
   if (threadIdx.x == 0) {
      theta = theta_;
      rung = rung_;
   }
}

CUDA_KERNEL cuda_kick_kernel(finite_vector<kick_return, KICK_GRID_SIZE> *rc,
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> *roots,
      finite_vector<int, KICK_GRID_SIZE> *depths, finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces) {
   const int &bid = blockIdx.x;

   (*rc)[bid] = cuda_kick((*roots)[bid], (*stacks)[bid], (*workspaces)[bid], (*depths)[bid]);

}

std::pair<std::function<bool()>, std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>>> cuda_execute_kick_kernel(
      finite_vector<kick_stack, KICK_GRID_SIZE> &&stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> &&roots,
      finite_vector<int, KICK_GRID_SIZE> &&depths, int grid_size) {
   std::vector < std::function < kick_return() >> returns;
   finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
   CUDA_MALLOC(rcptr, 1);
   new (rcptr) finite_vector<kick_return, KICK_GRID_SIZE>();
   rcptr->resize(grid_size);
   hipStream_t stream;
   hipEvent_t event;
   CUDA_CHECK(hipStreamCreate(&stream));
   CUDA_CHECK(hipEventCreate(&event));
   finite_vector<kick_workspace_t, KICK_GRID_SIZE> workspaces;
   workspaces.resize(KICK_GRID_SIZE);

   finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
   finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
   finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
   finite_vector<kick_workspace_t, KICK_GRID_SIZE>* workspaces_ptr;
   CUDA_MALLOC(stacks_ptr, 1);
   CUDA_MALLOC(roots_ptr, 1);
   CUDA_MALLOC(depths_ptr, 1);
   CUDA_MALLOC(workspaces_ptr, 1);
   new (stacks_ptr) finite_vector<kick_stack, KICK_GRID_SIZE>(std::move(stacks));
   new (roots_ptr) finite_vector<tree_ptr, KICK_GRID_SIZE>(std::move(roots));
   new (depths_ptr) finite_vector<int, KICK_GRID_SIZE>(std::move(depths));
   new (workspaces_ptr) finite_vector<kick_workspace_t, KICK_GRID_SIZE>(std::move(workspaces));

cuda_set_kick_params<<<1,1>>>(0.7,0);
   /***************************************************************************************************************************************************/
   /**/cuda_kick_kernel<<<grid_size, KICK_BLOCK_SIZE, 0, stream>>>(rcptr, stacks_ptr,roots_ptr, depths_ptr, workspaces_ptr);/**/
/**/            CUDA_CHECK(hipEventRecord(event, stream));/*******************************************************************************************************/
   /***************************************************************************************************************************************************/

   struct cuda_kick_future_shared {
      hipStream_t stream;
      hipEvent_t event;
      std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>> returns;
      finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
      int grid_size;
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
      finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
      finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
      finite_vector<kick_workspace_t, KICK_GRID_SIZE>* workspaces_ptr;
      mutable bool ready;
   public:
      cuda_kick_future_shared() {
         ready = false;
      }
      bool operator()() const {
         if (!ready) {
            if (hipEventQuery(event) == hipSuccess) {
               ready = true;
               CUDA_CHECK(hipStreamSynchronize(stream));
        //       printf( "Kernel done\n");
               CUDA_CHECK(hipEventDestroy(event));
               CUDA_CHECK(hipStreamDestroy(stream));
               *returns = std::move(*rcptr);
               rcptr->finite_vector<kick_return, KICK_GRID_SIZE>::~finite_vector<kick_return, KICK_GRID_SIZE>();
               CUDA_FREE(rcptr);
               stacks_ptr->finite_vector<kick_stack, KICK_GRID_SIZE>::~finite_vector<kick_stack, KICK_GRID_SIZE>();
               roots_ptr->finite_vector<tree_ptr, KICK_GRID_SIZE>::~finite_vector<tree_ptr, KICK_GRID_SIZE>();
               depths_ptr->finite_vector<int, KICK_GRID_SIZE>::~finite_vector<int, KICK_GRID_SIZE>();
               workspaces_ptr->finite_vector<kick_workspace_t, KICK_GRID_SIZE>::~finite_vector<kick_workspace_t, KICK_GRID_SIZE>();
               CUDA_FREE(stacks_ptr);
               CUDA_FREE(roots_ptr);
               CUDA_FREE(depths_ptr);
               CUDA_FREE(workspaces_ptr);
            }
         }
         return ready;
      }
   };

   cuda_kick_future_shared fut;
   fut.returns = std::make_shared<finite_vector<kick_return, KICK_GRID_SIZE>>();
   fut.stream = stream;
   fut.event = event;
   fut.rcptr = rcptr;
   fut.grid_size = grid_size;
   fut.stacks_ptr = stacks_ptr;
   fut.roots_ptr = roots_ptr;
   fut.depths_ptr = depths_ptr;
   fut.workspaces_ptr = workspaces_ptr;
   std::function < bool() > ready_func = [fut]() {
      return fut();
   };
   return std::make_pair(std::move(ready_func), std::move(fut.returns));
}

