#include "hip/hip_runtime.h"
#include <cosmictiger/tree.hpp>
#include <cosmictiger/cuda.hpp>
#include <functional>

//CUDA_KERNEL cuda_kick()

#define CC_CP_DIRECT 0
#define CC_CP_EWALD 1
#define PC_PP_DIRECT 2
#define PC_PP_EWALD 3
#define N_INTERACTION_TYPES 4

CUDA_DEVICE float theta;
CUDA_DEVICE int8_t rung;
CUDA_DEVICE particle_set *parts;

#define NITERS 4
#define MI 0
#define CI 1
#define OI 2
#define PI 3

struct cuda_kick_params {
   tree_ptr tptr;
   kick_stack &stacks;
   kick_workspace_t &workspace;
   int depth;
   CUDA_EXPORT cuda_kick_params(kick_stack &s, kick_workspace_t &w) :
         stacks(s), workspace(w) {
   }
};

using indices_array = array<array<int8_t, KICK_BLOCK_SIZE + 1>, NITERS>;
using counts_array = array<int16_t, NITERS>;

#define KICK_PP_MAX 128

using pos_array = array<fixed32,KICK_PP_MAX>;

#define MAX_BUCKET_SIZE 64
struct cuda_kick_shmem {
   indices_array indices;
   counts_array count;
   array<float, KICK_BLOCK_SIZE> f_x;
   array<float, KICK_BLOCK_SIZE> f_y;
   array<float, KICK_BLOCK_SIZE> f_z;
   array<float, MAX_BUCKET_SIZE> Fx;
   array<float, MAX_BUCKET_SIZE> Fy;
   array<float, MAX_BUCKET_SIZE> Fz;
   array<fixed32, KICK_PP_MAX> x;
   array<fixed32, KICK_PP_MAX> y;
   array<fixed32, KICK_PP_MAX> z;
};

CUDA_DEVICE void cuda_pp_interactions(cuda_kick_params &params) {
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   auto &f_x = shmem.f_x;
   auto &f_y = shmem.f_y;
   auto &f_z = shmem.f_z;
   auto &Fx = shmem.Fx;
   auto &Fy = shmem.Fy;
   auto &Fz = shmem.Fz;
   auto &source_x = shmem.x;
   auto &source_y = shmem.y;
   auto &source_z = shmem.z;
   const int &tid = threadIdx.x;
   auto &inters = params.workspace.part_interactions;
   if (inters.size()) {
      int i = 0;
      const auto &myparts = ((tree*) params.tptr)->parts;
      const size_t nsinks = myparts.second - myparts.first;
      __syncthreads();
      while (i < inters.size()) {
         pair<size_t, size_t> these_parts;
         these_parts = ((tree*) inters[i])->parts;
         i++;
         while (i < inters.size()) {
            auto next_parts = ((tree*) inters[i])->parts;
            if (next_parts.first == these_parts.second && next_parts.second - these_parts.first <= KICK_PP_MAX) {
               these_parts.second = ((tree*) inters[i])->parts.second;
               i++;
            } else {
               break;
            }
         }
         const auto offset = ((tree*) params.tptr)->parts.first;
         for (int j = these_parts.first + tid; j < these_parts.second; j += KICK_BLOCK_SIZE) {
            const auto j0 = j - these_parts.first;
            source_x[j0] = parts->pos(0, j);
            source_y[j0] = parts->pos(1, j);
            source_z[j0] = parts->pos(2, j);
         }
         __syncthreads();
         for (int k = 0; k < nsinks; k++) {
            f_x[tid] = f_y[tid] = f_x[tid];
            const auto sink_x = parts->pos(0, offset + k);
            const auto sink_y = parts->pos(1, offset + k);
            const auto sink_z = parts->pos(2, offset + k);
            for (int j = these_parts.first + tid; j < these_parts.second; j += KICK_BLOCK_SIZE) {
               const auto j0 = j - these_parts.first;
               const auto dx = (fixed<int32_t>(source_x[j0]) - fixed<int32_t>(sink_x)).to_float();
               const auto dy = (fixed<int32_t>(source_y[j0]) - fixed<int32_t>(sink_y)).to_float();
               const auto dz = (fixed<int32_t>(source_z[j0]) - fixed<int32_t>(sink_z)).to_float();
               const auto r2 = sqr(dx) + sqr(dy) + sqr(dz);
               const auto rinv = rsqrtf(r2);
               const auto rinv3 = rinv * rinv * rinv;
               f_x[tid] -= dx * rinv3;
               f_y[tid] -= dy * rinv3;
               f_z[tid] -= dz * rinv3;
            }
            for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
               if (tid < P) {
                  f_x[tid] += f_x[tid + P];
                  f_y[tid] += f_y[tid + P];
                  f_z[tid] += f_z[tid + P];
               }
               __syncthreads();
            }
            if (tid == 0) {
               Fx[k] += f_x[0];
               Fy[k] += f_y[0];
               Fz[k] += f_z[0];
            }
            __syncthreads();
         }
      }
   }
}

CUDA_DEVICE kick_return cuda_kick(cuda_kick_params &params) {
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   kick_stack &stacks = params.stacks;
   tree_ptr tptr = params.tptr;
   const int &tid = threadIdx.x;
   int depth = params.depth;
   kick_return rc;
   auto &Fx = shmem.Fx;
   auto &Fy = shmem.Fy;
   auto &Fz = shmem.Fz;
   if (((tree*) tptr)->children[0].rank == -1) {
      for (int k = tid; k < MAX_BUCKET_SIZE; k += KICK_BLOCK_SIZE) {
         Fx[k] = 0.f;
         Fy[k] = 0.f;
         Fz[k] = 0.f;
      }
      __syncthreads();
   }
   {
      indices_array &indices = shmem.indices;
      counts_array &count = shmem.count;

      kick_workspace_t &workspace = params.workspace;

      const auto theta2 = theta * theta;
      array<checks_type*, N_INTERACTION_TYPES> all_checks;
      all_checks[CC_CP_DIRECT] = &stacks.dchecks[depth];
      all_checks[CC_CP_EWALD] = &stacks.echecks[depth];
      all_checks[PC_PP_DIRECT] = &stacks.dchecks[depth];
      all_checks[PC_PP_EWALD] = &stacks.echecks[depth];
      array<finite_vector<tree_ptr, WORKSPACE_SIZE>*, NITERS> lists;
      auto &multis = workspace.multi_interactions;
      auto &parti = workspace.part_interactions;
      auto &next_checks = workspace.next_checks;
      auto &opened_checks = workspace.opened_checks;
      lists[MI] = &multis;
      lists[PI] = &parti;
      lists[CI] = &next_checks;
      lists[OI] = &opened_checks;
      const auto &myradius = ((tree*) tptr)->radius;
      const auto &mypos = ((tree*) tptr)->pos;
      int ninteractions = ((tree*) tptr)->children[0].rank == -1 ? 4 : 2;
      for (int type = 0; type < ninteractions; type++) {
         for (int i = 0; i < NITERS; i++) {
            lists[i]->resize(WORKSPACE_SIZE);
         }
         auto &checks = *(all_checks[type]);
         const bool ewald_dist = type == PC_PP_EWALD || type == CC_CP_EWALD;
         const bool direct = type == PC_PP_EWALD || type == PC_PP_DIRECT;
         if (tid < NITERS) {
            count[tid] = 0;
         }
         __syncthreads();
         int check_count;
         do {
            if (checks.size()) {
               check_count = checks.size();
               checks.resize(WORKSPACE_SIZE);
               const int cimax = ((check_count - 1) / KICK_BLOCK_SIZE + 1) * KICK_BLOCK_SIZE;
               for (int ci = tid; ci < cimax; ci += KICK_BLOCK_SIZE) {
                  for (int i = 0; i < NITERS; i++) {
                     indices[i][tid + 1] = 0;
                  }
                  __syncthreads();
                  if (tid < NITERS) {
                     indices[tid][0] = 0;
                  }
                  __syncthreads();
                  int list_index = -1;
                  if (ci < check_count) {
                     auto &check = checks[ci];
                     const auto &other_radius = ((const tree*) check)->radius;
                     const auto &other_pos = ((const tree*) check)->pos;
                     float d2 = 0.f;
                     const float R2 = sqr(other_radius + myradius);
                     for (int dim = 0; dim < NDIM; dim++) {
                        d2 += sqr(fixed<int32_t>(other_pos[dim]) - fixed<int32_t>(mypos[dim])).to_float();
                     }
                     if (ewald_dist) {
                        d2 = fmaxf(d2, EWALD_MIN_DIST2);
                     }
                     const bool far = R2 < theta2 * d2;
                     const bool isleaf = ((const tree*) check)->children[0].rank == -1;
                     list_index = int(!far) * (1 + int(isleaf) + int(isleaf && bool(check.opened++)));
                     indices[list_index][tid + 1] = 1;
                  }
                  __syncthreads();
                  for (int P = 1; P < KICK_BLOCK_SIZE; P *= 2) {
                     array<int, NITERS> tmp;
                     if (tid + 1 > P) {
                        for (int i = 0; i < NITERS; i++) {
                           tmp[i] = indices[i][tid - P + 1];
                        }
                     }
                     __syncthreads();
                     if (tid + 1 > P) {
                        for (int i = 0; i < NITERS; i++) {
                           indices[i][tid + 1] += tmp[i];
                        }
                     }
                     __syncthreads();
                  }
                  __syncthreads();
                  if (ci < check_count) {
                     const auto &check = checks[ci];
                     assert(count[list_index] + indices[list_index][tid] >= 0);
                     (*lists[list_index])[count[list_index] + indices[list_index][tid]] = check;
                  }
                  __syncthreads();
                  if (tid < NITERS) {
                     count[tid] += indices[tid][KICK_BLOCK_SIZE];
                  }
                  __syncthreads();
               }
               __syncthreads();
               for (int i = tid; i < count[CI]; i += KICK_BLOCK_SIZE) {
                  const auto children = next_checks[i].get_children();
                  for (int j = 0; j < NCHILD; j++) {
                     checks[2 * i + j] = children[j];
                  }
               }
               check_count = 2 * count[CI];
               if (type == CC_CP_DIRECT || type == CC_CP_EWALD) {
                  for (int i = tid; i < count[OI]; i += KICK_BLOCK_SIZE) {
                     checks[check_count + i] = opened_checks[i];
                  }
                  check_count += count[OI];
               } else {
                  for (int i = tid; i < count[OI]; i += KICK_BLOCK_SIZE) {
                     parti[count[PI] + i] = opened_checks[i];
                  }
               }
               __syncthreads();
               if (tid == 0) {
                  count[CI] = 0;
                  count[OI] = 0;
               }
               __syncthreads();
               checks.resize(check_count);
            }
         } while (direct && check_count);
         multis.resize(count[MI]);
         parti.resize(count[PI]);

         if (type == PC_PP_DIRECT) {
            cuda_pp_interactions(params);
         }

         /*********** DO INTERACTIONS *********************/

      }
   }
   if (!(((tree*) tptr)->children[0].rank == -1)) {
      stacks.dchecks[depth + 1] = stacks.dchecks[depth];
      stacks.echecks[depth + 1] = stacks.echecks[depth];
      params.depth++;
      params.tptr = ((tree*) tptr)->children[LEFT];
      kick_return rc1 = cuda_kick(params);
      stacks.dchecks[depth + 1] = std::move(stacks.dchecks[depth]);
      stacks.echecks[depth + 1] = std::move(stacks.echecks[depth]);
      params.tptr = ((tree*) tptr)->children[RIGHT];
      kick_return rc2 = cuda_kick(params);
      params.depth--;
      rc.rung = max(rc1.rung, rc2.rung);
   } else {
      rc.rung = 0;
   }
   return rc;
}

CUDA_KERNEL cuda_set_kick_params_kernel(particle_set *p, float theta_, int rung_) {
   if (threadIdx.x == 0) {
      parts = p;
      theta = theta_;
      rung = rung_;
   }
}

void tree::cuda_set_kick_params(particle_set *p, float theta_, int rung_) {
cuda_set_kick_params_kernel<<<1,1>>>(p,theta_,rung_);
                                                                                             CUDA_CHECK(hipDeviceSynchronize());
}

CUDA_KERNEL cuda_kick_kernel(finite_vector<kick_return, KICK_GRID_SIZE> *rc,
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> *roots,
      finite_vector<int, KICK_GRID_SIZE> *depths, finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces) {
   const int &bid = blockIdx.x;
   cuda_kick_params params((*stacks)[bid], (*workspaces)[bid]);
   params.tptr = (*roots)[bid];
   params.depth = (*depths)[bid];
   (*rc)[bid] = cuda_kick(params);

}

std::pair<std::function<bool()>, std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>>> cuda_execute_kick_kernel(
      finite_vector<kick_stack, KICK_GRID_SIZE> &&stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> &&roots,
      finite_vector<int, KICK_GRID_SIZE> &&depths, int grid_size) {
   std::vector < std::function < kick_return() >> returns;
   finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
   CUDA_MALLOC(rcptr, 1);
   new (rcptr) finite_vector<kick_return, KICK_GRID_SIZE>();
   rcptr->resize(grid_size);
   hipStream_t stream;
   hipEvent_t event;
   CUDA_CHECK(hipStreamCreate(&stream));
   CUDA_CHECK(hipEventCreate(&event));
   finite_vector<kick_workspace_t, KICK_GRID_SIZE> workspaces;
   workspaces.resize(KICK_GRID_SIZE);

   finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
   finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
   finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
   finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces_ptr;
   CUDA_MALLOC(stacks_ptr, 1);
   CUDA_MALLOC(roots_ptr, 1);
   CUDA_MALLOC(depths_ptr, 1);
   CUDA_MALLOC(workspaces_ptr, 1);
   //  printf("Shared mem requirements = %li\n", sizeof(cuda_kick_shmem));
   new (stacks_ptr) finite_vector<kick_stack, KICK_GRID_SIZE>(std::move(stacks));
   new (roots_ptr) finite_vector<tree_ptr, KICK_GRID_SIZE>(std::move(roots));
   new (depths_ptr) finite_vector<int, KICK_GRID_SIZE>(std::move(depths));
   new (workspaces_ptr) finite_vector<kick_workspace_t, KICK_GRID_SIZE>(std::move(workspaces));
   const size_t shmemsize = sizeof(cuda_kick_shmem);
   /***************************************************************************************************************************************************/
   /**/cuda_kick_kernel<<<grid_size, KICK_BLOCK_SIZE, shmemsize, stream>>>(rcptr, stacks_ptr,roots_ptr, depths_ptr, workspaces_ptr);/**/
   /**/   CUDA_CHECK(hipEventRecord(event, stream));/*******************************************************************************************************/
   /***************************************************************************************************************************************************/

   struct cuda_kick_future_shared {
      hipStream_t stream;
      hipEvent_t event;
      std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>> returns;
      finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
      int grid_size;
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
      finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
      finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
      finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces_ptr;
      mutable bool ready;
   public:
      cuda_kick_future_shared() {
         ready = false;
      }
      bool operator()() const {
         if (!ready) {
            if (hipEventQuery(event) == hipSuccess) {
               ready = true;
               CUDA_CHECK(hipStreamSynchronize(stream));
               CUDA_CHECK(hipEventDestroy(event));
               CUDA_CHECK(hipStreamDestroy(stream));
               *returns = std::move(*rcptr);
               rcptr->finite_vector<kick_return, KICK_GRID_SIZE>::~finite_vector<kick_return, KICK_GRID_SIZE>();
               stacks_ptr->finite_vector<kick_stack, KICK_GRID_SIZE>::~finite_vector<kick_stack, KICK_GRID_SIZE>();
               roots_ptr->finite_vector<tree_ptr, KICK_GRID_SIZE>::~finite_vector<tree_ptr, KICK_GRID_SIZE>();
               depths_ptr->finite_vector<int, KICK_GRID_SIZE>::~finite_vector<int, KICK_GRID_SIZE>();
               workspaces_ptr->finite_vector<kick_workspace_t, KICK_GRID_SIZE>::~finite_vector<kick_workspace_t,
               KICK_GRID_SIZE>();
               CUDA_FREE(rcptr);
               CUDA_FREE(stacks_ptr);
               CUDA_FREE(roots_ptr);
               CUDA_FREE(depths_ptr);
               CUDA_FREE(workspaces_ptr);
            }
         }
         return ready;
      }
   };

   cuda_kick_future_shared fut;
   fut.returns = std::make_shared<finite_vector<kick_return, KICK_GRID_SIZE>>();
   fut.stream = stream;
   fut.event = event;
   fut.rcptr = rcptr;
   fut.grid_size = grid_size;
   fut.stacks_ptr = stacks_ptr;
   fut.roots_ptr = roots_ptr;
   fut.depths_ptr = depths_ptr;
   fut.workspaces_ptr = workspaces_ptr;
   std::function < bool() > ready_func = [fut]() {
      return fut();
   };
   return std::make_pair(std::move(ready_func), std::move(fut.returns));
}

