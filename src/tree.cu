#include "hip/hip_runtime.h"
#include <cosmictiger/tree.hpp>
#include <cosmictiger/cuda.hpp>
#include <functional>

//CUDA_KERNEL cuda_kick()

#define CC_CP_DIRECT 0
#define CC_CP_EWALD 1
#define PC_PP_DIRECT 2
#define PC_PP_EWALD 3
#define N_INTERACTION_TYPES 4

CUDA_DEVICE float theta;
CUDA_DEVICE int8_t rung;
CUDA_DEVICE particle_set *parts;

#define NITERS 4
#define MI 0
#define CI 1
#define OI 2
#define PI 3

struct cuda_kick_params {
   tree_ptr tptr;
   kick_stack &stacks;
   kick_workspace_t &workspace;
   int depth;
   CUDA_EXPORT cuda_kick_params(kick_stack &s, kick_workspace_t &w) :
         stacks(s), workspace(w) {
   }
};

using indices_array = array<array<int8_t, KICK_BLOCK_SIZE + 1>, NITERS>;
using counts_array = array<int16_t, NITERS>;

using pos_array = array<fixed32,WORKSPACE_SIZE>;

CUDA_DEVICE void cuda_pp_interactions(cuda_kick_params &params) {
   const int &tid = threadIdx.x;
   const int &depth = params.depth;
   __shared__
   extern int shmem[];
   pos_array &xsource = *((pos_array*) (shmem));
   pos_array &ysource = *((pos_array*) ((int8_t*) shmem) + sizeof(pos_array));
   pos_array &zsource = *((pos_array*) ((int8_t*) shmem) + 2 * sizeof(pos_array));
   auto &inters = params.workspace.part_interactions;
   if (inters.size()) {
      int i = 0;
      while (i < inters.size()) {
         pair<size_t, size_t> these_parts;
         these_parts = ((tree*) inters[i])->parts;
         i++;
         while (i < inters.size()) {
            if (((tree*) inters[i])->parts.first == these_parts.second) {
               these_parts.second = ((tree*) inters[i])->parts.second;
               i++;
            } else {
               break;
            }
         }
         for (int j = these_parts.first + tid; j < these_parts.second; j += KICK_BLOCK_SIZE) {
            const auto offset = these_parts.first;
            assert(j - offset <= KICK_BLOCK_SIZE);
            xsource[j - offset] = parts->pos(0, j);
            ysource[j - offset] = parts->pos(1, j);
            zsource[j - offset] = parts->pos(2, j);
            printf("Loading\n");
         }
      }
   }
}

CUDA_DEVICE kick_return cuda_kick(cuda_kick_params &params) {
   __shared__
   extern int shmem[];
   kick_stack &stacks = params.stacks;
   tree_ptr tptr = params.tptr;
   const int &tid = threadIdx.x;
   int depth = params.depth;
   kick_return rc;
   {
      indices_array &indices = *(indices_array*) shmem;
      counts_array &count = *(counts_array*) (((uint8_t*) shmem) + sizeof(indices_array));

      kick_workspace_t &workspace = params.workspace;

      const auto theta2 = theta * theta;
      array<checks_type*, N_INTERACTION_TYPES> all_checks;
      all_checks[CC_CP_DIRECT] = &stacks.dchecks[depth];
      all_checks[CC_CP_EWALD] = &stacks.echecks[depth];
      all_checks[PC_PP_DIRECT] = &stacks.dchecks[depth];
      all_checks[PC_PP_EWALD] = &stacks.echecks[depth];
      array<finite_vector<tree_ptr, WORKSPACE_SIZE>*, NITERS> lists;
      auto &multis = workspace.multi_interactions;
      auto &parti = workspace.part_interactions;
      auto &next_checks = workspace.next_checks;
      auto &opened_checks = workspace.opened_checks;
      lists[MI] = &multis;
      lists[PI] = &parti;
      lists[CI] = &next_checks;
      lists[OI] = &opened_checks;
      const auto &myradius = ((tree*) tptr)->radius;
      const auto &mypos = ((tree*) tptr)->pos;
      int ninteractions = ((tree*) tptr)->children[0].rank == -1 ? 4 : 2;
      for (int type = 0; type < ninteractions; type++) {
         for (int i = 0; i < NITERS; i++) {
            lists[i]->resize(WORKSPACE_SIZE);
         }
         auto &checks = *(all_checks[type]);
         const bool ewald_dist = type == PC_PP_EWALD || type == CC_CP_EWALD;
         const bool direct = type == PC_PP_EWALD || type == PC_PP_DIRECT;
         if (tid < NITERS) {
            count[tid] = 0;
         }
         __syncthreads();
         int check_count;
         do {
            if (checks.size()) {
               check_count = checks.size();
               checks.resize(WORKSPACE_SIZE);
               const int cimax = ((check_count - 1) / KICK_BLOCK_SIZE + 1) * KICK_BLOCK_SIZE;
               for (int ci = tid; ci < cimax; ci += KICK_BLOCK_SIZE) {
                  for (int i = 0; i < NITERS; i++) {
                     indices[i][tid + 1] = 0;
                  }
                  __syncthreads();
                  if (tid < NITERS) {
                     indices[tid][0] = 0;
                  }
                  __syncthreads();
                  int list_index = -1;
                  if (ci < check_count) {
                     auto &check = checks[ci];
                     const auto &other_radius = ((const tree*) check)->radius;
                     const auto &other_pos = ((const tree*) check)->pos;
                     float d2 = 0.f;
                     const float R2 = sqr(other_radius + myradius);
                     for (int dim = 0; dim < NDIM; dim++) {
                        d2 += sqr(fixed<int32_t>(other_pos[dim]) - fixed<int32_t>(mypos[dim])).to_float();
                     }
                     if (ewald_dist) {
                        d2 = fmaxf(d2, EWALD_MIN_DIST2);
                     }
                     const bool far = R2 < theta2 * d2;
                     const bool isleaf = ((const tree*) check)->children[0].rank == -1;
                     list_index = int(!far) * (1 + int(isleaf) + int(isleaf && bool(check.opened++)));
                     indices[list_index][tid + 1] = 1;
                  }
                  __syncthreads();
                  for (int P = 1; P < KICK_BLOCK_SIZE; P *= 2) {
                     array<int, NITERS> tmp;
                     if (tid + 1 > P) {
                        for (int i = 0; i < NITERS; i++) {
                           tmp[i] = indices[i][tid - P + 1];
                        }
                     }
                     __syncthreads();
                     if (tid + 1 > P) {
                        for (int i = 0; i < NITERS; i++) {
                           indices[i][tid + 1] += tmp[i];
                        }
                     }
                     __syncthreads();
                  }
                  __syncthreads();
                  if (ci < check_count) {
                     const auto &check = checks[ci];
                     assert(count[list_index] + indices[list_index][tid] >= 0);
                     (*lists[list_index])[count[list_index] + indices[list_index][tid]] = check;
                  }
                  __syncthreads();
                  if (tid < NITERS) {
                     count[tid] += indices[tid][KICK_BLOCK_SIZE];
                  }
                  __syncthreads();
               }
               __syncthreads();
               for (int i = tid; i < count[CI]; i += KICK_BLOCK_SIZE) {
                  const auto children = next_checks[i].get_children();
                  for (int j = 0; j < NCHILD; j++) {
                     checks[2 * i + j] = children[j];
                  }
               }
               check_count = 2 * count[CI];
               if (type == CC_CP_DIRECT || type == CC_CP_EWALD) {
                  for (int i = tid; i < count[OI]; i += KICK_BLOCK_SIZE) {
                     checks[check_count + i] = opened_checks[i];
                  }
                  check_count += count[OI];
               } else {
                  for (int i = tid; i < count[OI]; i += KICK_BLOCK_SIZE) {
                     parti[count[PI] + i] = opened_checks[i];
                  }
               }
               __syncthreads();
               if (tid == 0) {
                  count[CI] = 0;
                  count[OI] = 0;
               }
               __syncthreads();
               checks.resize(check_count);
            }
         } while (direct && check_count);
         multis.resize(count[MI]);
         parti.resize(count[PI]);

         if (type == PC_PP_DIRECT) {
            //    cuda_pp_interactions(params);
         }

         /*********** DO INTERACTIONS *********************/

      }
   }
   if (!(((tree*) tptr)->children[0].rank == -1)) {
      stacks.dchecks[depth + 1] = stacks.dchecks[depth];
      stacks.echecks[depth + 1] = stacks.echecks[depth];
      params.depth++;
      params.tptr = ((tree*) tptr)->children[LEFT];
      kick_return rc1 = cuda_kick(params);
      stacks.dchecks[depth + 1] = std::move(stacks.dchecks[depth]);
      stacks.echecks[depth + 1] = std::move(stacks.echecks[depth]);
      params.tptr = ((tree*) tptr)->children[RIGHT];
      kick_return rc2 = cuda_kick(params);
      params.depth--;
      rc.rung = max(rc1.rung, rc2.rung);
   } else {
      rc.rung = 0;
   }
   return rc;
}

CUDA_KERNEL cuda_set_kick_params_kernel(particle_set *p, float theta_, int rung_) {
   if (threadIdx.x == 0) {
      parts = p;
      theta = theta_;
      rung = rung_;
   }
}

void tree::cuda_set_kick_params(particle_set *p, float theta_, int rung_) {
cuda_set_kick_params_kernel<<<1,1>>>(p,theta_,rung_);
                        CUDA_CHECK(hipDeviceSynchronize());
}

CUDA_KERNEL cuda_kick_kernel(finite_vector<kick_return, KICK_GRID_SIZE> *rc,
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> *roots,
      finite_vector<int, KICK_GRID_SIZE> *depths, finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces) {
   const int &bid = blockIdx.x;
   cuda_kick_params params((*stacks)[bid], (*workspaces)[bid]);
   params.tptr = (*roots)[bid];
   params.depth = (*depths)[bid];
   (*rc)[bid] = cuda_kick(params);

}

std::pair<std::function<bool()>, std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>>> cuda_execute_kick_kernel(
      finite_vector<kick_stack, KICK_GRID_SIZE> &&stacks, finite_vector<tree_ptr, KICK_GRID_SIZE> &&roots,
      finite_vector<int, KICK_GRID_SIZE> &&depths, int grid_size) {
   std::vector < std::function < kick_return() >> returns;
   finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
   CUDA_MALLOC(rcptr, 1);
   new (rcptr) finite_vector<kick_return, KICK_GRID_SIZE>();
   rcptr->resize(grid_size);
   hipStream_t stream;
   hipEvent_t event;
   CUDA_CHECK(hipStreamCreate(&stream));
   CUDA_CHECK(hipEventCreate(&event));
   finite_vector<kick_workspace_t, KICK_GRID_SIZE> workspaces;
   workspaces.resize(KICK_GRID_SIZE);

   finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
   finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
   finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
   finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces_ptr;
   CUDA_MALLOC(stacks_ptr, 1);
   CUDA_MALLOC(roots_ptr, 1);
   CUDA_MALLOC(depths_ptr, 1);
   CUDA_MALLOC(workspaces_ptr, 1);
   new (stacks_ptr) finite_vector<kick_stack, KICK_GRID_SIZE>(std::move(stacks));
   new (roots_ptr) finite_vector<tree_ptr, KICK_GRID_SIZE>(std::move(roots));
   new (depths_ptr) finite_vector<int, KICK_GRID_SIZE>(std::move(depths));
   new (workspaces_ptr) finite_vector<kick_workspace_t, KICK_GRID_SIZE>(std::move(workspaces));
   const size_t shmemsize = std::max(sizeof(indices_array) + sizeof(counts_array), NDIM * sizeof(pos_array));
   /***************************************************************************************************************************************************/
   /**/cuda_kick_kernel<<<grid_size, KICK_BLOCK_SIZE, shmemsize, stream>>>(rcptr, stacks_ptr,roots_ptr, depths_ptr, workspaces_ptr);/**/
   /**/   CUDA_CHECK(hipEventRecord(event, stream));/*******************************************************************************************************/
   /***************************************************************************************************************************************************/

   struct cuda_kick_future_shared {
      hipStream_t stream;
      hipEvent_t event;
      std::shared_ptr<finite_vector<kick_return, KICK_GRID_SIZE>> returns;
      finite_vector<kick_return, KICK_GRID_SIZE> *rcptr;
      int grid_size;
      finite_vector<kick_stack, KICK_GRID_SIZE> *stacks_ptr;
      finite_vector<tree_ptr, KICK_GRID_SIZE> *roots_ptr;
      finite_vector<int, KICK_GRID_SIZE> *depths_ptr;
      finite_vector<kick_workspace_t, KICK_GRID_SIZE> *workspaces_ptr;
      mutable bool ready;
   public:
      cuda_kick_future_shared() {
         ready = false;
      }
      bool operator()() const {
         if (!ready) {
            if (hipEventQuery(event) == hipSuccess) {
               ready = true;
               CUDA_CHECK(hipStreamSynchronize(stream));
               CUDA_CHECK(hipEventDestroy(event));
               CUDA_CHECK(hipStreamDestroy(stream));
               *returns = std::move(*rcptr);
               rcptr->finite_vector<kick_return, KICK_GRID_SIZE>::~finite_vector<kick_return, KICK_GRID_SIZE>();
               stacks_ptr->finite_vector<kick_stack, KICK_GRID_SIZE>::~finite_vector<kick_stack, KICK_GRID_SIZE>();
               roots_ptr->finite_vector<tree_ptr, KICK_GRID_SIZE>::~finite_vector<tree_ptr, KICK_GRID_SIZE>();
               depths_ptr->finite_vector<int, KICK_GRID_SIZE>::~finite_vector<int, KICK_GRID_SIZE>();
               workspaces_ptr->finite_vector<kick_workspace_t, KICK_GRID_SIZE>::~finite_vector<kick_workspace_t,
               KICK_GRID_SIZE>();
               CUDA_FREE(rcptr);
               CUDA_FREE(stacks_ptr);
               CUDA_FREE(roots_ptr);
               CUDA_FREE(depths_ptr);
               CUDA_FREE(workspaces_ptr);
            }
         }
         return ready;
      }
   };

   cuda_kick_future_shared fut;
   fut.returns = std::make_shared<finite_vector<kick_return, KICK_GRID_SIZE>>();
   fut.stream = stream;
   fut.event = event;
   fut.rcptr = rcptr;
   fut.grid_size = grid_size;
   fut.stacks_ptr = stacks_ptr;
   fut.roots_ptr = roots_ptr;
   fut.depths_ptr = depths_ptr;
   fut.workspaces_ptr = workspaces_ptr;
   std::function < bool() > ready_func = [fut]() {
      return fut();
   };
   return std::make_pair(std::move(ready_func), std::move(fut.returns));
}

