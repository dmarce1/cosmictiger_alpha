#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <cosmictiger/particle.hpp>
#include <cosmictiger/global.hpp>
#include <cosmictiger/memory.hpp>
#include <cosmictiger/fixed.hpp>

#define BLOCK_SIZE 32
#define COUNT_BLOCKS 92

CUDA_KERNEL morton_keygen(particle::flags_t *flags, fixed32 *xptr, fixed32 *yptr, fixed32 *zptr, size_t nele,
      size_t depth) {
   const int &tid = threadIdx.x;
   const int &bid = blockIdx.x;
   const size_t shift = (sizeof(fixed32) * CHAR_BIT - depth / NDIM);
   const size_t start = bid * nele / gridDim.x;
   const size_t stop = (bid + 1) * nele / gridDim.x;
   for (size_t i = start + tid; i < stop; i += BLOCK_SIZE) {
      morton_t key = 0LL;
      size_t x[NDIM];
      x[0] = xptr[i].get_integer() >> shift;
      x[1] = yptr[i].get_integer() >> shift;
      x[2] = zptr[i].get_integer() >> shift;
      for (size_t k = 0; k < depth / NDIM; k++) {
         for (size_t dim = 0; dim < NDIM; dim++) {
            key ^= size_t((bool) (x[dim] & (1LL << k))) << size_t(k * NDIM + (NDIM - 1 - dim));
         }
      }
      //     printf( "%lx\n",key);
      flags[i].morton_id = key;
      //   printf( "%i\n", nele);
   }
}

CUDA_KERNEL count_keys(int *counts, particle::flags_t *keys, morton_t key_min, morton_t key_max, size_t nele) {
   const int &tid = threadIdx.x;
   const int &bid = blockIdx.x;
   const size_t start = bid * nele / gridDim.x;
   const size_t stop = (bid + 1) * nele / gridDim.x;
   for (size_t i = start + tid; i < stop; i += BLOCK_SIZE) {
      const size_t index = keys[i].morton_id - key_min;
//      if(keys[i].morton_id < key_min ) {
//         printf( "min out %lx %lx\n", keys[i].morton_id, key_min);
//         __trap();
//      }
//      if(keys[i].morton_id >= key_max ) {
//         printf( "max out %lx %lx\n", keys[i].morton_id, key_max);
//         __trap();
//      }
      assert(keys[i].morton_id >= key_min);
      assert(keys[i].morton_id < key_max);
      atomicAdd(counts + index, 1);
   }
}

std::pair<std::vector<size_t>, std::vector<size_t>> cuda_keygen(particle_set &set, size_t start, size_t stop, int depth,
      morton_t key_min, morton_t key_max) {
   const int nblocks = (92 * 32 - 1) / BLOCK_SIZE + 1;
   int *counts;
   const size_t size = key_max - key_min;
   start -= set.offset_;
   stop -= set.offset_;
   assert(stop > start);
   fixed32 *x = set.xptr_[0] + start;
   fixed32 *y = set.xptr_[1] + start;
   fixed32 *z = set.xptr_[2] + start;
   particle::flags_t *flags = set.rptr_ + start;
morton_keygen<<<nblocks, BLOCK_SIZE>>>(flags,x,y,z,stop-start, depth);
                     CUDA_CHECK(hipDeviceSynchronize());
   // printf( "KEYS         %lx %lx %lx %lx \n", key_min, *key_min, *key_max, key_stop);

   CUDA_MALLOC(counts, size);
   for (int i = 0; i < size; i++) {
      counts[i] = 0;
   }
count_keys<<<COUNT_BLOCKS,BLOCK_SIZE>>>(counts,  flags, key_min, key_max, stop - start);
                     CUDA_CHECK(hipDeviceSynchronize());
//            printf( "%li %li %li %li  %li  \n", key_min, *key_min, *key_max, key_stop, key_stop - key_min + 1 >= *key_max - *key_min) ;

   std::pair<std::vector<size_t>, std::vector<size_t>> bounds;
   bounds.first.resize(key_max - key_min + 1);
   bounds.second.resize(key_max - key_min + 1);

   bounds.first[0] = start;
   for (int i = 1; i <= size; i++) {
      bounds.first[i] = counts[i - 1] + bounds.first[i - 1];
   }
   CUDA_FREE(counts);
   for (int i = 1; i <= size; i++) {
      bounds.second[i - 1] = bounds.first[i];
   }
   return std::move(bounds);
}

//
//CUDA_KERNEL radix_sort_count(size_t *count, morton_t *keys, morton_t key_min, morton_t key_max) {
//
//}
//
//
//CUDA_KERNEL radix_sort_do_sort(fixed32 *x, fixed32 *y, fixed32 *z, fixed32 *vx, fixed32 *vy, fixed32 *vz, rung_t *rung,
//      size_t *begin, size_t *end, morton_t key_min, size_t nele) {
//
//}
