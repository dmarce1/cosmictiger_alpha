#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <cosmictiger/particle.hpp>
#include <cosmictiger/global.hpp>
#include <cosmictiger/memory.hpp>
#include <cosmictiger/fixed.hpp>

#define BLOCK_SIZE 32
#define COUNT_BLOCKS 92

CUDA_KERNEL morton_keygen(particle::flags_t *flags, morton_t *key_min, morton_t *key_max, fixed32 *xptr, fixed32 *yptr,
      fixed32 *zptr, size_t nele, size_t depth) {
   const int &tid = threadIdx.x + blockDim.x * BLOCK_SIZE;
   const int &bid = blockIdx.x;
   const size_t shift = (sizeof(fixed32) * CHAR_BIT - depth / NDIM);
   const size_t start = bid * nele / gridDim.x;
   const size_t stop = (bid + 1) * nele / gridDim.x;
   for (size_t i = start + tid; i < stop; i += BLOCK_SIZE) {
      morton_t key = 0LL;
      size_t x[NDIM];
      x[0] = xptr[i].get_integer() >> shift;
      x[1] = yptr[i].get_integer() >> shift;
      x[2] = zptr[i].get_integer() >> shift;
      for (size_t k = 0; k < depth / NDIM; k++) {
         for (size_t dim = 0; dim < NDIM; dim++) {
            key ^= size_t((bool) (x[dim] & (0x0000000000000001LL << k))) << size_t(k * NDIM + dim);
         }
      }
      //     printf( "%lx\n",key);
      *key_min = atomicMin((unsigned long long*) key_min, (unsigned long long) key);
      *key_max = atomicMax((unsigned long long*) key_max, (unsigned long long) key);
      flags[i].morton_id = key;
      //   printf( "%i\n", nele);
      __syncthreads();
   }
}

CUDA_KERNEL count_keys(int *counts, particle::flags_t *keys, morton_t key_min, morton_t key_max, size_t nele) {
   const int &tid = threadIdx.x + blockDim.x * BLOCK_SIZE;
   const int &bid = blockIdx.x;
   const size_t start = bid * nele / gridDim.x;
   const size_t stop = (bid + 1) * nele / gridDim.x;
   for (size_t i = start + tid; i < stop; i += BLOCK_SIZE) {
      const size_t index = keys[i].morton_id - key_min;
      atomicAdd(counts + 0, 1);
   }
}

std::vector<size_t> cuda_keygen(particle_set &set, size_t start, size_t stop, int depth) {
   morton_t *key_min;
   morton_t *key_max;
   CUDA_MALLOC(key_min, 1);
   CUDA_MALLOC(key_max, 1);
   *key_min = ~(1 << (depth + 1));
   *key_max = 0;
   start -= set.offset_;
   stop -= set.offset_;
   fixed32 *x = set.xptr_[0] + start;
   fixed32 *y = set.xptr_[1] + start;
   fixed32 *z = set.xptr_[2] + start;
   particle::flags_t *flags = set.rptr_ + start;
//
   const int nblocks = (92 * 32 - 1) / BLOCK_SIZE + 1;
morton_keygen<<<nblocks, BLOCK_SIZE>>>(flags,key_min,key_max,x,y,z,stop-start, depth);
         CUDA_CHECK(hipDeviceSynchronize());
  //
       //  printf("-----%li %li\n", *key_min, *key_max);
   int *counts;
   (*key_max)++;
   const size_t size = *key_max - *key_min;
 //  printf( "-------%li\n", size);
   CUDA_MALLOC(counts, size);
   for (int i = 0; i < *key_max - *key_min; i++) {
      counts[i] = 0;
   }
count_keys<<<COUNT_BLOCKS,BLOCK_SIZE>>>(counts,  flags, *key_min, *key_max, stop-start);
         CUDA_CHECK(hipDeviceSynchronize());
   std::vector < size_t > bounds(*key_max - *key_min);
   bounds[0] = start + set.offset_;
   for (size_t i = 1; i < *key_max - *key_min; i++) {
      bounds[i] = bounds[i - 1] + counts[i - 1];
   }

   CUDA_FREE(counts);
   CUDA_FREE(key_max);
   CUDA_FREE(key_min);

   return bounds;
}

//
//CUDA_KERNEL radix_sort_count(size_t *count, morton_t *keys, morton_t key_min, morton_t key_max) {
//
//}
//
//
//CUDA_KERNEL radix_sort_do_sort(fixed32 *x, fixed32 *y, fixed32 *z, fixed32 *vx, fixed32 *vy, fixed32 *vz, rung_t *rung,
//      size_t *begin, size_t *end, morton_t key_min, size_t nele) {
//
//}
