#include <cosmictiger/particle.hpp>
#include <cosmictiger/math.hpp>
#include <cosmictiger/global.hpp>

void particle_set::generate_random(int seed) {

	if (size_) {
		hipFuncAttributes attribs;
		CUDA_CHECK(hipFuncGetAttributes(&attribs, reinterpret_cast<const void*>(generate_random_vectors)));
		int num_threads = attribs.maxThreadsPerBlock;
		int num_blocks;
		CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, generate_random_vectors, num_threads, 0));
		num_blocks *= global().cuda.devices[0].multiProcessorCount;
		printf( "%i x %i\n", num_blocks,num_threads);
		generate_random_vectors<<<num_blocks,num_threads>>>(xptr_[0],xptr_[1],xptr_[2],global().opts.nparts,seed);
		CUDA_CHECK(hipDeviceSynchronize());

		for (int i = 0; i < size_; i++) {
			for (int dim = 0; dim < NDIM; dim++) {
				vel(0, i) = 0.f;
				vel(1, i) = 0.f;
				vel(2, i) = 0.f;
			}
			set_rung(0, i);
		}
	}
}
