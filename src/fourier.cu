#include "hip/hip_runtime.h"
#include <cosmictiger/fourier.hpp>
#include <cosmictiger/global.hpp>
#include <hipfft/hipfft.h>

#define FFTSIZE_COMPUTE 32
#define FFTSIZE_TRANSPOSE 32

/* error checker from https://forums.developer.nvidia.com/t/cufft-error-handling/29231 */
static const char *_cudaGetErrorEnum(hipfftResult error) {
	switch (error) {
	case HIPFFT_SUCCESS:
		return "HIPFFT_SUCCESS";

	case HIPFFT_INVALID_PLAN:
		return "HIPFFT_INVALID_PLAN";

	case HIPFFT_ALLOC_FAILED:
		return "HIPFFT_ALLOC_FAILED";

	case HIPFFT_INVALID_TYPE:
		return "HIPFFT_INVALID_TYPE";

	case HIPFFT_INVALID_VALUE:
		return "HIPFFT_INVALID_VALUE";

	case HIPFFT_INTERNAL_ERROR:
		return "HIPFFT_INTERNAL_ERROR";

	case HIPFFT_EXEC_FAILED:
		return "HIPFFT_EXEC_FAILED";

	case HIPFFT_SETUP_FAILED:
		return "HIPFFT_SETUP_FAILED";

	case HIPFFT_INVALID_SIZE:
		return "HIPFFT_INVALID_SIZE";

	case HIPFFT_UNALIGNED_DATA:
		return "HIPFFT_UNALIGNED_DATA";
	}

	return "<unknown>";
}

inline void _cuda_fft_check(hipfftResult err, const char *file, const int line) {
	if (HIPFFT_SUCCESS != err) {
		fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n", __FILE__, __LINE__, err,
				_cudaGetErrorEnum(err));
		hipDeviceReset();
		assert(0);
	}
}

#define CUDA_FFT_CHECK(a) _cuda_fft_check(a,__FILE__,__LINE__)

__global__
void transpose_2d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;
	for (int xi = bid; xi < N; xi += grid_size) {
		for (int yi = tid; yi < N; yi += block_size) {
			if (xi < yi) {
				const int i1 = (N * xi + yi);
				const int i2 = (N * yi + xi);
				const cmplx tmp = Y[i1];
				Y[i1] = Y[i2];
				Y[i2] = tmp;
			}
		}
	}
}

__global__
void normalize_invert_2d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;
	const float N3inv = 1.0f / (N * sqr(N));
	for (int xi = bid; xi < N; xi += grid_size) {
		for (int yi = tid; yi < N; yi += block_size) {
			const int i1 = (N * xi + yi);
			Y[i1].real() *= N3inv;
			Y[i1].imag() *= N3inv;
		}
	}

}
__global__
void transpose_xy_3d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;

	for (int xy = bid; xy < N * N; xy += grid_size) {
		int xi = xy / N;
		int yi = xy % N;
		if (xi < yi) {
			for (int zi = tid; zi < N; zi += block_size) {
				const int i1 = N * (N * xi + yi) + zi;
				const int i2 = N * (N * yi + xi) + zi;
				const cmplx tmp = Y[i1];
				Y[i1] = Y[i2];
				Y[i2] = tmp;
			}
		}
	}

}

__global__
void normalize_invert_3d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;
	const float N3inv = 1.0f / (N * sqr(N));
	for (int xy = bid; xy < N * N; xy += grid_size) {
		int xi = xy / N;
		int yi = xy % N;
		for (int zi = tid; zi < N; zi += block_size) {
			const int i1 = N * (N * xi + yi) + zi;
			Y[i1].real() *= N3inv;
			Y[i1].imag() *= N3inv;
		}
	}

}

__global__
void transpose_xz_3d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;
	for (int xy = bid; xy < N * N; xy += grid_size) {
		int xi = xy / N;
		int yi = xy % N;
		for (int zi = tid; zi < xi; zi += block_size) {
			const int i1 = N * (N * xi + yi) + zi;
			const int i2 = N * (N * zi + yi) + xi;
			const cmplx tmp = Y[i1];
			Y[i1] = Y[i2];
			Y[i2] = tmp;
		}
	}
}

__global__
void transpose_yz_3d(cmplx* Y, int N) {
	const int& tid = threadIdx.x;
	const int block_size = blockDim.x;
	const int& bid = blockIdx.x;
	const int& grid_size = gridDim.x;
	for (int xy = bid; xy < N * N; xy += grid_size) {
		int xi = xy / N;
		int yi = xy % N;
		for (int zi = tid; zi < yi; zi += block_size) {
			const int i1 = N * (N * xi + yi) + zi;
			const int i2 = N * (N * xi + zi) + yi;
			const cmplx tmp = Y[i1];
			Y[i1] = Y[i2];
			Y[i2] = tmp;
		}
	}
}

void fft3d(cmplx* Y, int N) {
	cuda_set_device();
	const int maxgrid = global().cuda.devices[0].maxGridSize[0];
	int nblocksc = min(N * N * N / FFTSIZE_COMPUTE, maxgrid);
	int nblockst = min(N * N * N / FFTSIZE_TRANSPOSE, maxgrid);
	hipfftHandle plan;
	CUDA_FFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, N * N));
	CUDA_FFT_CHECK(hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD));
	transpose_yz_3d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_FFT_CHECK(hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD));
	transpose_xz_3d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_FFT_CHECK(hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD));
	transpose_yz_3d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	transpose_xy_3d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	CUDA_CHECK(hipDeviceSynchronize());
}

void fft3d_inv(cmplx* Y, int N) {
	cuda_set_device();
	const int maxgrid = global().cuda.devices[0].maxGridSize[0];
	int nblocks = min(N * N * N / 32, maxgrid);
	normalize_invert_3d<<<nblocks,32>>>(Y,N);
	fft3d(Y, N);
}

void fft2d(cmplx* Y, int N) {
	cuda_set_device();
	const int maxgrid = global().cuda.devices[0].maxGridSize[0];
	int nblocksc = min(N * N / FFTSIZE_COMPUTE, maxgrid);
	int nblockst = min(N * N / FFTSIZE_TRANSPOSE, maxgrid);
	hipfftHandle plan;
	CUDA_FFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, N));
	CUDA_FFT_CHECK(hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD));
	transpose_2d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_FFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, N));
	transpose_2d<<<nblockst,FFTSIZE_TRANSPOSE>>>(Y,N);
	CUDA_CHECK(hipDeviceSynchronize());
}

void fft1d(cmplx* Y, int N) {
	cuda_set_device();
	const int maxgrid = global().cuda.devices[0].maxGridSize[0];
	int nblocksc = min(N * N / FFTSIZE_COMPUTE, maxgrid);
	int nblockst = min(N * N / FFTSIZE_TRANSPOSE, maxgrid);
	hipfftHandle plan;
	CUDA_FFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, N));
	CUDA_FFT_CHECK(hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD));
	CUDA_CHECK(hipDeviceSynchronize());
}

void fft32_inv(cmplx* Y, int N) {
	cuda_set_device();
	const int maxgrid = global().cuda.devices[0].maxGridSize[0];
	int nblocks = min(N * N / 32, maxgrid);
	normalize_invert_2d<<<nblocks,32>>>(Y,N);
	fft2d(Y, N);
}
