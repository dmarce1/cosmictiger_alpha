#include "hip/hip_runtime.h"
#include <cosmictiger/math.hpp>

#include <hiprand/hiprand_kernel.h>

double find_root(std::function<double(double)> f) {
	double x = 0.5;
	double err;
	int iters = 0;
	do {
		double dx0 = x * 1.0e-6;
		if (abs(dx0) == 0.0) {
			dx0 = 1.0e-10;
		}
		double fx = f(x);
		double dfdx = (f(x + dx0) - fx) / dx0;
		double dx = -fx / dfdx;
		err = abs(dx / max(1.0, abs(x)));
		x += 0.5 * dx;
		iters++;
		if (iters > 1000000) {
			printf("Finished early with error = %e\n", err);
			break;
		}
	} while (err > 1.0e-6);
	return x;
}

__global__
void generate_random_normals(cmplx* nums, int N, int seed) {
	const uint64_t mod = 1LL << 31LL;
	const uint64_t a1 = 1664525LL;
	const uint64_t a2 = 22695477LL;
	const uint64_t a3 = 134775813LL;
	const uint64_t a4 = 214013LL;
	const uint64_t a5 = 16843009LL;
	const uint64_t c1 = 1013904223LL;
	const uint64_t c2 = 1LL;
	const uint64_t c3 = 1LL;
	const uint64_t c4 = 2531011LL;
	const uint64_t c5 = 826366247LL;
	const int& thread = threadIdx.x;
	const int& block_size = blockDim.x;
	const auto count = threadIdx.x + blockIdx.x * blockDim.x;
	uint64_t int1 = seed;
	uint64_t int2 = (a1 * seed + c1) % mod;
	for (int i = 0; i < count; i++) {
		int1 = (a2 * int1 + c2) % mod;
		int2 = (a3 * int2 + c3) % mod;
	}
	const int start = blockIdx.x * N / gridDim.x;
	const int stop = (blockIdx.x + 1) * N / gridDim.x;
	for (int i = start + thread; i < stop; i += block_size) {
		int1 = (a4 * (uint64_t) int1 + c4) % mod;
		int2 = (a5 * (uint64_t) int2 + c5) % mod;
		const float x = ((float) int1 + 0.5f) / (float) uint64_t(mod + uint64_t(1));
		const float y1 = ((float) int2 + 0.5f) / (float) uint64_t(mod + uint64_t(1));
		const float y = 2.f * (float) M_PI * y1;
		nums[i] = sqrtf(-logf(fabsf(x))) * expc(cmplx(0, 1) * y);
	}
	__syncthreads();
}
