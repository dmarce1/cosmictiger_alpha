#include "hip/hip_runtime.h"
/*
 * gravity.cu
 *
 *  Created on: Feb 10, 2021
 *      Author: dmarce1
 */

#include <cosmictiger/gravity.hpp>

CUDA_DEVICE void cuda_cc_interactions(particle_set *parts, kick_params_type *params_ptr) {
   kick_params_type &params = *params_ptr;
   const int &tid = threadIdx.x;
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   auto &flops = shmem.flops;
   auto &Lreduce = shmem.Lreduce;
   auto &multis = params.multi_interactions;
   for (int i = 0; i < LP; i++) {
      Lreduce[tid][i] = 0.0;
   }
   __syncthreads();
   const auto &pos = ((tree*) params.tptr)->pos;
   for (int i = tid; i < params.nmulti; i += KICK_BLOCK_SIZE) {
      const multipole mpole = *((tree*) multis[i])->multi;
      expansion<float> L;
      array<float, NDIM> fpos;
      for (int dim = 0; dim < NDIM; dim++) {
         fpos[dim] = (fixed<int32_t>(pos[dim]) - fixed<int32_t>(pos[dim])).to_float();
      }
      flops[tid] += NDIM;
      flops[tid] += multipole_interaction(L, mpole, fpos, false);
      for (int j = 0; j < LP; j++) {
         Lreduce[tid][j] += L[j];
      }
      flops[tid] += LP;
   }
   __syncthreads();
   for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
      if (tid < P) {
         for (int i = 0; i < LP; i++) {
            Lreduce[tid][i] += Lreduce[tid + P][i];
         }
         flops[tid] += LP;
      }
      __syncthreads();
   }
   for (int i = tid; i < LP; i += KICK_BLOCK_SIZE) {
      params.L[params.depth][i] += Lreduce[0][i];
      flops[tid]++;
   }
}

CUDA_DEVICE int cuda_ewald_cc_interactions(particle_set *parts, kick_params_type *params_ptr) {
   kick_params_type &params = *params_ptr;
   const int &tid = threadIdx.x;
   __shared__
   extern int shmem_ptr[];
   cuda_ewald_shmem &shmem = *(cuda_ewald_shmem*) shmem_ptr;
   auto &flops = shmem.flops;
   auto &Lreduce = shmem.Lreduce;
   auto &multis = params.multi_interactions;
   for (int i = 0; i < LP; i++) {
      Lreduce[tid][i] = 0.0;
   }
   flops[tid] = 0;
   __syncthreads();
   const auto &pos = ((tree*) params.tptr)->pos;
   for (int i = tid; i < params.nmulti; i += KICK_BLOCK_SIZE) {
      const multipole mpole_float = *((tree*) multis[i])->multi;
      multipole_type<ewald_real> mpole;
      for (int i = 0; i < MP; i++) {
         mpole[i] = mpole_float[i];
      }
      expansion<ewald_real> L;
      array<ewald_real, NDIM> fpos;
      for (int dim = 0; dim < NDIM; dim++) {
#ifdef EWALD_DOUBLE_PRECISION
         fpos[dim] = (fixed<int32_t>(pos[dim]) - fixed<int32_t>(pos[dim])).to_double();
#else
         fpos[dim] = (fixed<int32_t>(pos[dim]) - fixed<int32_t>(pos[dim])).to_float();
#endif
      }
      flops[tid] += 3;
      flops[tid] += multipole_interaction_ewald(L, mpole, fpos, false);
      for (int j = 0; j < LP; j++) {
         Lreduce[tid][j] += L[j];
      }
      flops[tid] += 17;
   }
   __syncthreads();
   for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
      if (tid < P) {
         for (int i = 0; i < LP; i++) {
            Lreduce[tid][i] += Lreduce[tid + P][i];
            flops[tid]++;
         }
      }
      __syncthreads();
   }
   for (int i = tid; i < LP; i += KICK_BLOCK_SIZE) {
      params.L[params.depth][i] += Lreduce[0][i];
      flops[0]++;
   }
   return flops[0];
}

CUDA_DEVICE void cuda_cp_interactions(particle_set *parts, kick_params_type *params_ptr) {
   kick_params_type &params = *params_ptr;
   const int &tid = threadIdx.x;
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   auto &flops = shmem.flops;
   auto &Lreduce = shmem.Lreduce;
   auto &inters = params.part_interactions;
   const auto &sinks = ((tree*) params.tptr)->pos;
   auto &sources = shmem.src;
   const auto &myparts = ((tree*) params.tptr)->parts;
   size_t part_index;
   int i = 0;
   __syncthreads();
   if (params.npart > 0) {
      auto these_parts = ((tree*) inters[0])->parts;
      while (i < params.npart) {
         part_index = 0;
         while (part_index < KICK_PP_MAX && i < params.npart) {
            while (i + 1 < params.npart) {
               if (these_parts.second == ((tree*) inters[i + 1])->parts.first) {
                  these_parts.second = ((tree*) inters[i + 1])->parts.second;
                  i++;
               } else {
                  break;
               }
            }
            const size_t imin = these_parts.first;
            const size_t imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
            for (size_t j = imin + tid; j < imax; j += KICK_BLOCK_SIZE) {
               for (int dim = 0; dim < NDIM; dim++) {
                  sources[dim][part_index + j - imin] = parts->pos(dim, j);
               }
            }
            these_parts.first += imax - imin;
            part_index += imax - imin;
            if (these_parts.first == these_parts.second) {
               i++;
               if (i < params.npart) {
                  these_parts = ((tree*) inters[i])->parts;
               }
            }
         }
         for (int j = these_parts.first + tid; j < these_parts.second; j += KICK_BLOCK_SIZE) {
            array<float, NDIM> dx;
            for (int dim = 0; dim < NDIM; dim++) {
               dx[dim] = (fixed<int32_t>(parts->pos(dim, j)) - fixed<int32_t>(sinks[dim])).to_float();
            }
            flops[tid] += NDIM;
            expansion<float> L;
            flops[tid] += multipole_interaction(L, 1.0f, dx, false);
            for (int j = 0; j < LP; j++) {
               Lreduce[tid][j] += L[j];
            }
            flops[tid] += LP;
         }
         __syncthreads();
         for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
            if (tid < P) {
               for (int i = 0; i < LP; i++) {
                  Lreduce[tid][i] += Lreduce[tid + P][i];
               }
               flops[tid] += LP;
            }
            __syncthreads();
         }
         for (int i = tid; i < LP; i += KICK_BLOCK_SIZE) {
            params.L[params.depth][i] += Lreduce[0][i];
            flops[tid]++;
         }
      }
   }
}

CUDA_DEVICE void cuda_pp_interactions(particle_set *parts, kick_params_type *params_ptr) {
   kick_params_type &params = *params_ptr;
   const int &tid = threadIdx.x;
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   auto &f = shmem.f;
   auto &F = shmem.F;
#ifdef COUNT_FLOPS
   auto &flops = shmem.flops;
#endif
   auto &sources = shmem.src;
   auto &sinks = shmem.sink;
   auto &inters = params.part_interactions;
   const auto h2 = sqr(params.hsoft);
   size_t part_index;
   if (params.npart) {
      const auto &myparts = ((tree*) params.tptr)->parts;
      const size_t nsinks = myparts.second - myparts.first;
      for (int i = tid; i < nsinks; i += KICK_BLOCK_SIZE) {
         const auto this_rung = parts->rung(i + myparts.first);
         if (this_rung >= params.rung || this_rung == -1) {
            for (int dim = 0; dim < NDIM; dim++) {
               sinks[dim][i] = parts->pos(dim, i + myparts.first);
            }
         }
      }
      int i = 0;
      __syncthreads();
      auto these_parts = ((tree*) inters[0])->parts;
      while (i < params.npart) {
         part_index = 0;
         while (part_index < KICK_PP_MAX && i < params.npart) {
            while (i + 1 < params.npart) {
               if (these_parts.second == ((tree*) inters[i + 1])->parts.first) {
                  these_parts.second = ((tree*) inters[i + 1])->parts.second;
                  i++;
               } else {
                  break;
               }
            }
            const size_t imin = these_parts.first;
            const size_t imax = min(these_parts.first + (KICK_PP_MAX - part_index), these_parts.second);
            for (size_t j = imin + tid; j < imax; j += KICK_BLOCK_SIZE) {
               for (int dim = 0; dim < NDIM; dim++) {
                  sources[dim][part_index + j - imin] = parts->pos(dim, j);
               }
            }
            these_parts.first += imax - imin;
            part_index += imax - imin;
            if (these_parts.first == these_parts.second) {
               i++;
               if (i < params.npart) {
                  these_parts = ((tree*) inters[i])->parts;
               }
            }
         }
         __syncthreads();
         const auto offset = ((tree*) params.tptr)->parts.first;
         for (int k = 0; k < nsinks; k++) {
            const auto this_rung = parts->rung(k + offset);
#ifndef TEST_FORCE
            if (this_rung >= params.rung || this_rung == -1) {
#endif
               for (int dim = 0; dim < NDIM; dim++) {
                  f[dim][tid] = 0.f;
               }
               for (int j = tid; j < part_index; j += KICK_BLOCK_SIZE) {
                  array<float, NDIM> dx;
                  for (int dim = 0; dim < NDIM; dim++) { // 3
                     dx[dim] = (fixed<int32_t>(sources[dim][j]) - fixed<int32_t>(sinks[dim][k])).to_float();
                  }
                  const auto r2 = sqr(dx[0]) + sqr(dx[1]) + sqr(dx[2]); // 5
                  const auto rinv = rsqrtf(fmaxf(r2, h2)); // 8
                  const auto rinv3 = rinv * rinv * rinv; // 2
                  for (int dim = 0; dim < NDIM; dim++) { // 6
                     f[dim][tid] -= dx[dim] * rinv3;
                  }
#ifdef COUNT_FLOPS
                  flops[tid] += 24;
#endif
#ifndef TEST_FORCE
               }
#endif
               __syncthreads();
               for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
                  if (tid < P) {
                     for (int dim = 0; dim < NDIM; dim++) {
                        f[dim][tid] += f[dim][tid + P];
#ifdef COUNT_FLOPS
                        flops[tid]++;
#endif
                     }
                  }
                  __syncthreads();
               }
               if (tid == 0) {
                  for (int dim = 0; dim < NDIM; dim++) {
                     F[dim][k] += f[dim][0];
#ifdef COUNT_FLOPS
                     flops[tid]++;
#endif
                  }
               }
               __syncthreads();
            }
         }
      }
   }
}

CUDA_DEVICE
void cuda_pc_interactions(particle_set *parts, kick_params_type *params_ptr) {
   kick_params_type &params = *params_ptr;
   const int &tid = threadIdx.x;
   __shared__
   extern int shmem_ptr[];
   cuda_kick_shmem &shmem = *(cuda_kick_shmem*) shmem_ptr;
   auto &flops = shmem.flops;
   auto &f = shmem.f;
   auto &F = shmem.F;
   auto &sinks = shmem.sink;
   auto &inters = params.multi_interactions;
   const auto &myparts = ((tree*) params.tptr)->parts;
   const auto offset = myparts.first;
   const int mmax = ((params.nmulti - 1) / KICK_BLOCK_SIZE + 1) * KICK_BLOCK_SIZE;
   const int nparts = myparts.second - myparts.first;
   for (int i = tid; i < nparts; i += KICK_BLOCK_SIZE) {
      const auto this_rung = parts->rung(i + myparts.first);
      if (this_rung >= params.rung || this_rung == -1) {
         for (int dim = 0; dim < NDIM; dim++) {
            sinks[dim][i] = parts->pos(dim, myparts.first + i);
         }
      }
   }
   for (int i = tid; i < mmax; i += KICK_BLOCK_SIZE) {
      const auto &sources = ((tree*) inters[i])->pos;
      const int nparts = myparts.second - myparts.first;
      for (int k = 0; k < nparts; k++) {
         const auto this_rung = parts->rung(k + offset);
#ifndef TEST_FORCE
         if (this_rung >= params.rung || this_rung == -1) {
#endif
            for (int dim = 0; dim < NDIM; dim++) {
               f[dim][tid] = 0.f;
            }
            __syncthreads();
            if (i < params.nmulti) {
               array<float, NDIM> dx;
               array<float, NDIM + 1> Lforce;
               for (int l = 0; l < NDIM + 1; l++) {
                  Lforce[l] = 0.f;
               }
               for (int dim = 0; dim < NDIM; dim++) {
                  dx[dim] = (fixed<int32_t>(sources[dim]) - fixed<int32_t>(sinks[dim][k])).to_float();
               }
               flops[tid] += NDIM;
               flops[tid] += multipole_interaction(Lforce, *((tree*) inters[i])->multi, dx, false);
               for (int dim = 0; dim < NDIM; dim++) {
                  f[dim][tid] -= Lforce[dim + 1];
               }
               flops[tid] += NDIM;
            }
            __syncthreads();
            for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
               if (tid < P) {
                  for (int dim = 0; dim < NDIM; dim++) {
                     f[dim][tid] += f[dim][tid + P];
                  }
                  flops[tid] += NDIM;
               }
               __syncthreads();
            }
            if (tid == 0) {
               for (int dim = 0; dim < NDIM; dim++) {
                  F[dim][k] += f[dim][0];
               }
               flops[tid] += NDIM;
            }
            __syncthreads();
#ifndef TEST_FORCE
         }
#endif
      }
   }
}

#ifdef TEST_FORCE

CUDA_DEVICE extern ewald_indices *four_indices_ptr;
CUDA_DEVICE extern ewald_indices *real_indices_ptr;
CUDA_DEVICE extern periodic_parts *periodic_parts_ptr;

CUDA_KERNEL cuda_pp_ewald_interactions(particle_set *parts, size_t *test_parts, array<float, NDIM> *res) {
   const int &tid = threadIdx.x;
   const int &bid = blockIdx.x;
   const auto &hparts = *periodic_parts_ptr;
   const auto &four_indices = *four_indices_ptr;
   const auto &real_indices = *real_indices_ptr;

   const auto index = test_parts[bid];
   const auto src_x = parts->pos(0, index).to_float();
   const auto src_y = parts->pos(1, index).to_float();
   const auto src_z = parts->pos(2, index).to_float();
   __shared__ array<array<float, NDIM>, KICK_BLOCK_SIZE>
   f;
   for (int dim = 0; dim < NDIM; dim++) {
      f[dim][tid] = 0.0;
   }
   for (size_t sink = tid; sink < parts->size(); sink += KICK_BLOCK_SIZE) {
      if (sink == index) {
         continue;
      }
      array<float, NDIM> X;
      X[0] = src_x;
      X[1] = src_y;
      X[2] = src_z;
      for (int i = 0; i < real_indices.size(); i++) {
         const auto n = real_indices.get(i);
         array<ewald_real, NDIM> dx;
         for( int dim = 0; dim < NDIM; dim++) {
            dx[dim] = X[dim] - n[dim];
         }
         const float r2 = sqr(dx[0]) + sqr(dx[1]) + sqr(dx[2]);
         if (r2 < (EWALD_REAL_CUTOFF * EWALD_REAL_CUTOFF)) {  // 1
            const float r = sqrt(r2);  // 1
            const float cmask = 1.f - ((sqr(n[0])+sqr(n[1])+sqr(n[2])) > 0.0);  // 7
            const float rinv = 1.f / r;  // 2
            const float r2inv = rinv * rinv;  // 1
            const float r3inv = r2inv * rinv;  // 1
            const float exp0 = expf(-4.f * r2);  // 26
            const float erfc0 = erfcf(2.f * r);                                    // 10
            const float expfactor = 4.0 / sqrtf(M_PI) * r * exp0;  // 2
            const float e1 = expfactor * r3inv;  // 1
            const float d0 = -erfc0 * rinv;  // 2
            const float d1 = fma(-d0, r2inv, e1);  // 3
            for (int dim = 0; dim < NDIM; dim++) {
               f[dim][tid] = dx[dim] * d1;
            }
         }
      }
      for (int i = 0; i < four_indices.size(); i++) {
         const auto &h = four_indices.get(i);
         const auto &hpart = hparts.get(i);
         const float h2 = sqrt(h[0]) + sqr(h[1]) + sqr(h[2]);
         const float hdotx = h[0] * X[0] + h[1] * X[1] + h[2] * X[2];
         float so = sinf(2.0 * M_PI * hdotx);
         for (int dim = 0; dim < NDIM; dim++) {
            f[dim][tid] -= hpart(dim) * so;
         }
      }
   }
   __syncthreads();
   for (int P = KICK_BLOCK_SIZE / 2; P >= 1; P /= 2) {
      if (tid < P) {
         for (int dim = 0; dim < NDIM; dim++) {
            f[dim][tid] += f[dim][tid + P];
         }
      }
      __syncthreads();
   }
   for (int dim = 0; dim < NDIM; dim++) {
      res[index][dim] = f[dim][0];
   }
}

#endif
