#include "hip/hip_runtime.h"
#include <cosmictiger/sort.hpp>
#include <cosmictiger/global.hpp>
#include <cosmictiger/particle.hpp>

#define COUNT_BLOCK_SIZE 1024
#define SORT_BLOCK_SIZE 32
#define SORT_OCCUPANCY (16*128)

CUDA_KERNEL count_kernel(particle_set parts, size_t begin, size_t end, fixed32 xpos, int xdim, size_t* counts) {
	const int& tid = threadIdx.x;
	const int& bsz = blockDim.x;
	const int& bid = blockIdx.x;
	const int& gsz = gridDim.x;
	const size_t start = begin + bid * (end - begin) / gsz;
	const size_t stop = begin + (bid + 1) * (end - begin) / gsz;
	__shared__ array<size_t, COUNT_BLOCK_SIZE> local_counts;
	size_t my_count = 0;
	for (size_t i = start + tid; i < stop; i += bsz) {
		if (parts.pos(xdim, i) < xpos) {
			my_count++;
		}
	}
	local_counts[tid] = my_count;
	__syncthreads();
	for (int P = COUNT_BLOCK_SIZE / 2; P >= 1; P /= 2) {
		if (tid < P) {
			local_counts[tid] += local_counts[tid + P];
		}
		__syncthreads();
	}
	counts[bid] = local_counts[0];
}

CUDA_KERNEL sort_kernel(particle_set parts, size_t begin, size_t end, fixed32 xmid, int xdim,
		unsigned long long* bottom) {
	const int& tid = threadIdx.x;
	const int& bsz = blockDim.x;
	const int& bid = blockIdx.x;
	const int& gsz = gridDim.x;
	const size_t mid = (begin + end) / 2;
	const size_t start = mid + bid * (end - mid) / gsz;
	const size_t stop = mid + (bid + 1) * (end - mid) / gsz;
	for (size_t i = tid + start; i < stop; i += SORT_BLOCK_SIZE) {
		bool found_swap = (parts.pos(xdim, i) < xmid);
		while (__any_sync(0xFFFFFFFF, found_swap)) {
			int my_index = found_swap;
			int count = found_swap;
			for (int P = SORT_BLOCK_SIZE / 2; P >= 1; P /= 2) {
				count += __shfl_down_sync(0xFFFFFFFF, count, P);
			}
			for (int P = 1; P < SORT_BLOCK_SIZE; P *= 2) {
				int tmp = __shfl_up_sync(0xFFFFFFFF, my_index, P);
				if (tid >= P) {
					my_index += tmp;
				}
			}
			my_index = __shfl_up_sync(0xFFFFFFFF, my_index, 1);
			size_t base_index;
			if (tid == 0) {
				my_index = 0;
				base_index = atomicAdd(bottom, count);
			}
			base_index = __shfl_sync(-1, base_index, 0);
			size_t swap_index = base_index + my_index;
			if (found_swap) {
				if (!(parts.pos(xdim, swap_index) < xmid)) {
					parts.swap(i, swap_index);
					found_swap = false;
				}
			}
		}
	}
}

size_t count_particles(particle_set parts, size_t begin, size_t end, fixed32 xpos, int xdim) {
	const auto nparts = global().opts.nparts;
	const auto nprocs = global().cuda.devices[0].multiProcessorCount;
	const auto mycount = end - begin;
	int nchunks = std::max(1, (int) (mycount * nprocs / nparts));
	size_t* counts;
	CUDA_MALLOC(counts, nchunks);
	auto stream = get_stream();
	parts.prepare_sort1(xdim, stream);
	count_kernel<<<nchunks,COUNT_BLOCK_SIZE,0,stream>>>(parts,begin,end,xpos,xdim,counts);
	CUDA_CHECK(hipStreamSynchronize(stream));
	cleanup_stream(stream);
	size_t count = 0;
	for (int i = 0; i < nchunks; i++) {
		count += counts[i];
	}
	CUDA_FREE(counts);
	return count;
}

fixed32 find_median(particle_set parts, size_t begin, size_t end, fixed32 xmin, fixed32 xmax, int xdim) {
	int64_t half = (end - begin) / 2;
	int64_t lastmid;
	int64_t countmid = 0;
	int64_t countmax;
	fixed32 xmid;
	bool first_call = true;
	do {
		xmid = fixed32((fixed64(xmin) + fixed64(xmax)) / fixed64(2));
		lastmid = countmid;
		countmid = (int64_t) count_particles(parts, begin, end, xmid, xdim) - half;
		if (first_call) {
			countmax = (int64_t) count_particles(parts, begin, end, xmax, xdim) - half;
			first_call = false;
		}
//		printf("%li %li %e %e %e \n", countmid, half, xmin.to_float(), xmid.to_float(), xmax.to_float());
		if (countmid * countmax < 0) {
			xmin = xmid;
		} else {
			xmax = xmid;
			countmax = countmid;
		}
	} while (lastmid != countmid && countmid);
	unsigned long long* bottom;
	CUDA_MALLOC(bottom, 1);
	*bottom = 0;
	const auto nparts = global().opts.nparts;
	const auto nprocs = global().cuda.devices[0].multiProcessorCount;
	const auto mycount = end - begin;
	int nchunks = std::max(1, (int) (mycount * nprocs / nparts));
	printf("Sorting %i\n", nchunks);
	auto stream = get_stream();
	parts.prepare_sort2(stream);
	sort_kernel<<<nchunks*SORT_OCCUPANCY,SORT_BLOCK_SIZE,0,stream>>>(parts, begin, end, xmid, xdim, bottom);
	CUDA_CHECK(hipStreamSynchronize(stream));
	cleanup_stream(stream);
	printf("Done Sorting\n");
	CUDA_FREE(bottom);
	return xmid;
}
