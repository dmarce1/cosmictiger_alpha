#include "hip/hip_runtime.h"
#include <cosmictiger/initial.hpp>
#include <cosmictiger/fourier.hpp>
#include <cosmictiger/global.hpp>
#include <cosmictiger/boltzmann.hpp>
#include <cosmictiger/zero_order.hpp>
#include <cosmictiger/zeldovich.hpp>
#include <cosmictiger/constants.hpp>

#define BOLTZSIZE 256
#define FFTSIZE 256
#define RANDSIZE 256
#define ZELDOSIZE 256

template<class T>
__global__ void vector_free_kernel(vector<T>* vect) {
	if (threadIdx.x == 0) {
		vect->vector < T > ::~vector<T>();
	}
}

void initial_conditions(particle_set& parts) {

	int numBlocks;
	sigma8_integrand *func_ptr;
	zero_order_universe* zeroverse_ptr;
	float* result_ptr;
	cos_state* states;
	cosmic_params params;
	int Nk = 100;
	interp_functor<float>* den_k;
	interp_functor<float>* vel_k;
	cmplx* basis;
	cmplx* phi;
	cmplx* rands;
	const size_t N = global().opts.parts_dim;
	const size_t N3 = sqr(N) * N;

	CUDA_MALLOC(phi, N3);
	CUDA_MALLOC(rands, N3);
	CUDA_MALLOC(basis, N / 2);
	CUDA_MALLOC(den_k, 1);
	CUDA_MALLOC(vel_k, 1);
	CUDA_MALLOC(zeroverse_ptr, 1);
	CUDA_MALLOC(result_ptr, 1);
	CUDA_MALLOC(func_ptr, 1);
	CUDA_MALLOC(states, Nk);

	new (den_k) interp_functor<float>();
	new (vel_k) interp_functor<float>();

	auto& uni = *zeroverse_ptr;

#ifndef __CUDA_ARCH__
	auto den_destroy = den_k->to_device();
	auto vel_destroy = vel_k->to_device();
	auto cs_destroy = uni.cs2.to_device();
	auto sigma_destroy = uni.sigma_T.to_device();
#endif
	params.omega_b = global().opts.omega_b;
	params.omega_c = global().opts.omega_c;
	params.omega_gam = global().opts.omega_gam;
	params.omega_nu = global().opts.omega_nu;
	params.Y = global().opts.Y;
	params.Neff = global().opts.Neff;
	params.Theta = global().opts.Theta;
	params.hubble = global().opts.hubble;
	printf("Computing zero order universe...");
	fflush(stdout);
	create_zero_order_universe(&uni, 1.0e6, params);
	set_zeroverse(&uni);
	printf("Done.\n");
	func_ptr->uni = zeroverse_ptr;
	func_ptr->littleh = params.hubble;
	printf("Computing sigma8 normalization...");
	fflush(stdout);
	float kmin = (1e-4 * params.hubble);
	float kmax = (5 * params.hubble);
	integrate<sigma8_integrand, float> <<<1, BOLTZSIZE>>>(func_ptr,
			(float) std::log(kmin), (float) std::log(kmax), result_ptr, (float) 1.0e-6);
	CUDA_CHECK(hipDeviceSynchronize());
	*result_ptr = sqrt(sqr(global().opts.sigma8) / *result_ptr);
	printf("Done. Normalization = %e\n", *result_ptr);
	float normalization = *result_ptr;

	int block_size = min(BOLTZSIZE, Nk);
/*		printf("\tComputing Einstain-Boltzmann interpolation solutions for power.dat\n");
	 float dk = log(kmax / kmin) / (Nk - 1);
	 printf("\tComputing Einstain-Boltzmann interpolation solutions for wave numbers %e to %e Mpc^-1\n", kmin, kmax);
	 einstein_boltzmann_interpolation_function<<<1, block_size>>>(den_k, vel_k, states, zeroverse_ptr, kmin, kmax, normalization, Nk, zeroverse_ptr->amin, 1.f);
	 CUDA_CHECK(hipDeviceSynchronize());
*/
	const auto code_to_mpc = global().opts.code_to_cm / constants::mpc_to_cm;
	printf( "code_to_mpc = %e\n", code_to_mpc);

	kmin = 2.0 * (float) M_PI / code_to_mpc;
	kmax = sqrtf(3) * (kmin * (float) (global().opts.parts_dim));
	printf("\tComputing Einstain-Boltzmann interpolation solutions for wave numbers %e to %e Mpc^-1\n", kmin, kmax);
	Nk = 2 * global().opts.parts_dim;
	block_size = min(BOLTZSIZE, Nk);
	const float ainit = 1.0f / (global().opts.z0 + 1.0f);
	einstein_boltzmann_interpolation_function<<<1, block_size>>>(den_k, vel_k, states, zeroverse_ptr, kmin, kmax, normalization, Nk, zeroverse_ptr->amin, ainit);
	CUDA_CHECK(hipDeviceSynchronize());

	printf("\tComputing FFT basis\n");
	fft_basis<<<1,FFTSIZE>>>(basis, N);
	CUDA_CHECK(hipDeviceSynchronize());

	printf("\tComputing random number set\n");
	generate_random_normals<<<1,RANDSIZE>>>(rands, N3);
	CUDA_CHECK(hipDeviceSynchronize());

	printf("\tComputing over/under density\n");
	zeldovich<<<1,ZELDOSIZE>>>(phi, basis, rands, den_k, code_to_mpc, N, 0, DENSITY);
	CUDA_CHECK(hipDeviceSynchronize());
	fft3d(phi, basis, N);
	float drho = 0.0;
	for (int i = 0; i < N3; i++) {
		drho = std::max(drho, std::abs((phi[i].real())));
	}
	printf("\t\tOver/under density is %e\n", drho);

	float xdisp = 0.0, vmax = 0.0;
	const double omega_m = params.omega_b + params.omega_c;
	const double omega_r = params.omega_nu + params.omega_gam;
	const double a = ainit;
	const double Om = omega_m / (omega_m + (a * a * a) * (1.0 - omega_m - omega_r) + omega_r / a);
	const double f = std::pow(Om, 0.55);
	const double H = global().opts.H0
			* std::sqrt(omega_r / (a * a * a * a) + omega_m / (a * a * a) + 1.0 - omega_r - omega_m);
	const double prefac = f * H * a;
	printf("Velocity prefactor is %e, Hubble(a) = %e, f(a) = %e\n", prefac, H, f);
	for (int dim = 0; dim < NDIM; dim++) {
		printf("\t\tComputing %c positions\n", 'x' + dim);
		zeldovich<<<1,ZELDOSIZE>>>(phi, basis, rands, den_k, code_to_mpc, N, dim, DISPLACEMENT);
		CUDA_CHECK(hipDeviceSynchronize());
		fft3d(phi, basis, N);
		for (int i = 0; i < N3; i++) {
			xdisp = std::max(xdisp, std::abs((phi[i].real())));
		}
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < N; k++) {
					const int l = N * (N * i + j) + k;
					const int I[NDIM] = { i, j, k };
					float x = (((float) I[dim] + 0.5f) / (float) N);
					x += phi[l].real() / code_to_mpc;
					while (x > 1.0) {
						x -= 1.0;
					}
					while (x < 0.0) {
						x += 1.0;
					}
					parts.pos(dim, l) = x;
				}
			}
		}
		printf("\t\tComputing %c velocities\n", 'x' + dim);
		zeldovich<<<1,ZELDOSIZE>>>(phi, basis, rands, den_k, code_to_mpc, N, dim, VELOCITY);
		CUDA_CHECK(hipDeviceSynchronize());
		fft3d(phi, basis, N);
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < N; k++) {
					const int l = N * (N * i + j) + k;
					float v = phi[l].real();
					parts.vel(l).a[dim] = v * a / code_to_mpc;
				}
			}
		}
	}
	xdisp /= code_to_mpc / N;
	printf("\t\tMaximum displacement is %e\n", xdisp);

#ifndef __CUDA_ARCH__
	den_destroy();
	vel_destroy();
	cs_destroy();
	sigma_destroy();
#endif
	vector_free_kernel<<<1,1>>>(&vel_k->values);
	vector_free_kernel<<<1,1>>>(&den_k->values);
	vector_free_kernel<<<1,1>>>(&uni.sigma_T.values);
	vector_free_kernel<<<1,1>>>(&uni.cs2.values);
	CUDA_FREE(zeroverse_ptr);
	CUDA_FREE(result_ptr);
	CUDA_FREE(func_ptr);
	CUDA_FREE(states);
	CUDA_FREE(vel_k);
	CUDA_FREE(den_k);
	CUDA_FREE(basis);
	CUDA_FREE(rands);
	CUDA_FREE(phi);
}

