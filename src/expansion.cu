#include "hip/hip_runtime.h"
/*
 * expansion.cpp
 *
 *  Created on: Feb 10, 2021
 *      Author: dmarce1
 */

#include <cosmictiger/expansion.hpp>
#include <cosmictiger/array.hpp>

__device__ expansion<float> Lfactor_gpu;
expansion<float> Lfactor_cpu;

__device__ void expansion_init() {
	for (int i = 0; i < LP; i++) {
		Lfactor_gpu[i] = float(0.0);
	}
	Lfactor_gpu() += float(1);
	for (int a = 0; a < NDIM; ++a) {
		Lfactor_gpu(a) += float(1.0);
		for (int b = 0; b < NDIM; ++b) {
			Lfactor_gpu(a, b) += float(0.5);
			for (int c = 0; c < NDIM; ++c) {
				Lfactor_gpu(a, b, c) += float(1.0 / 6.0);
				for (int d = 0; d < NDIM; ++d) {
					Lfactor_gpu(a, b, c, d) += float(1.0 / 24.0);
				}
			}
		}
	}
}

__host__ void expansion_init_cpu() {
	for (int i = 0; i < LP; i++) {
		Lfactor_cpu[i] = float(0.0);
	}
	Lfactor_cpu() += float(1);
	for (int a = 0; a < NDIM; ++a) {
		Lfactor_cpu(a) += float(1.0);
		for (int b = 0; b < NDIM; ++b) {
			Lfactor_cpu(a, b) += float(0.5);
			for (int c = 0; c < NDIM; ++c) {
				Lfactor_cpu(a, b, c) += float(1.0 / 6.0);
				for (int d = 0; d < NDIM; ++d) {
					Lfactor_cpu(a, b, c, d) += float(1.0 / 24.0);
				}
			}
		}
	}
	// for( int i = 0; i < 35; i++) {
	// 	printf( "%e\n", 1.0/Lfactor_cpu[i]);
	//}
	//abort();
}

CUDA_EXPORT expansion<float>& shift_expansion(expansion<float> &L, const array<float, NDIM> &dX) {
	float tmp1, tmp2;
	L[0] = fma(L[1], dX[0], L[0]);
	tmp1 = dX[0] * dX[0];
	L[0] = fma(L[4], tmp1 * float(5.000000e-01), L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[10], tmp2 * float(1.666667e-01), L[0]);
	L[0] = fma(L[20], tmp2 * dX[0] * float(4.166667e-02), L[0]);
	L[0] = fma(L[2], dX[1], L[0]);
	tmp1 = dX[1] * dX[0];
	L[0] = fma(L[5], tmp1, L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[11], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[21], tmp2 * dX[0] * float(1.666667e-01), L[0]);
	tmp1 = dX[1] * dX[1];
	L[0] = fma(L[7], tmp1 * float(5.000000e-01), L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[13], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[23], tmp2 * dX[0] * float(2.500000e-01), L[0]);
	tmp2 = tmp1 * dX[1];
	L[0] = fma(L[16], tmp2 * float(1.666667e-01), L[0]);
	L[0] = fma(L[26], tmp2 * dX[0] * float(1.666667e-01), L[0]);
	L[0] = fma(L[30], tmp2 * dX[1] * float(4.166667e-02), L[0]);
	L[0] = fma(L[3], dX[2], L[0]);
	tmp1 = dX[2] * dX[0];
	L[0] = fma(L[6], tmp1, L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[12], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[22], tmp2 * dX[0] * float(1.666667e-01), L[0]);
	tmp1 = dX[2] * dX[1];
	L[0] = fma(L[8], tmp1, L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[14], tmp2, L[0]);
	L[0] = fma(L[24], tmp2 * dX[0] * float(5.000000e-01), L[0]);
	tmp2 = tmp1 * dX[1];
	L[0] = fma(L[17], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[27], tmp2 * dX[0] * float(5.000000e-01), L[0]);
	L[0] = fma(L[31], tmp2 * dX[1] * float(1.666667e-01), L[0]);
	tmp1 = dX[2] * dX[2];
	L[0] = fma(L[9], tmp1 * float(5.000000e-01), L[0]);
	tmp2 = tmp1 * dX[0];
	L[0] = fma(L[15], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[25], tmp2 * dX[0] * float(2.500000e-01), L[0]);
	tmp2 = tmp1 * dX[1];
	L[0] = fma(L[18], tmp2 * float(5.000000e-01), L[0]);
	L[0] = fma(L[28], tmp2 * dX[0] * float(5.000000e-01), L[0]);
	L[0] = fma(L[32], tmp2 * dX[1] * float(2.500000e-01), L[0]);
	tmp2 = tmp1 * dX[2];
	L[0] = fma(L[19], tmp2 * float(1.666667e-01), L[0]);
	L[0] = fma(L[29], tmp2 * dX[0] * float(1.666667e-01), L[0]);
	L[0] = fma(L[33], tmp2 * dX[1] * float(1.666667e-01), L[0]);
	L[0] = fma(L[34], tmp2 * dX[2] * float(4.166667e-02), L[0]);
	L[1] = fma(L[4], dX[0], L[1]);
	tmp1 = dX[0] * dX[0];
	L[1] = fma(L[10], tmp1 * float(5.000000e-01), L[1]);
	L[1] = fma(L[20], tmp1 * dX[0] * float(1.666667e-01), L[1]);
	L[1] = fma(L[5], dX[1], L[1]);
	tmp1 = dX[1] * dX[0];
	L[1] = fma(L[11], tmp1, L[1]);
	L[1] = fma(L[21], tmp1 * dX[0] * float(5.000000e-01), L[1]);
	tmp1 = dX[1] * dX[1];
	L[1] = fma(L[13], tmp1 * float(5.000000e-01), L[1]);
	L[1] = fma(L[23], tmp1 * dX[0] * float(5.000000e-01), L[1]);
	L[1] = fma(L[26], tmp1 * dX[1] * float(1.666667e-01), L[1]);
	L[1] = fma(L[6], dX[2], L[1]);
	tmp1 = dX[2] * dX[0];
	L[1] = fma(L[12], tmp1, L[1]);
	L[1] = fma(L[22], tmp1 * dX[0] * float(5.000000e-01), L[1]);
	tmp1 = dX[2] * dX[1];
	L[1] = fma(L[14], tmp1, L[1]);
	L[1] = fma(L[24], tmp1 * dX[0], L[1]);
	L[1] = fma(L[27], tmp1 * dX[1] * float(5.000000e-01), L[1]);
	tmp1 = dX[2] * dX[2];
	L[1] = fma(L[15], tmp1 * float(5.000000e-01), L[1]);
	L[1] = fma(L[25], tmp1 * dX[0] * float(5.000000e-01), L[1]);
	L[1] = fma(L[28], tmp1 * dX[1] * float(5.000000e-01), L[1]);
	L[1] = fma(L[29], tmp1 * dX[2] * float(1.666667e-01), L[1]);
	L[2] = fma(L[5], dX[0], L[2]);
	tmp1 = dX[0] * dX[0];
	L[2] = fma(L[11], tmp1 * float(5.000000e-01), L[2]);
	L[2] = fma(L[21], tmp1 * dX[0] * float(1.666667e-01), L[2]);
	L[2] = fma(L[7], dX[1], L[2]);
	tmp1 = dX[1] * dX[0];
	L[2] = fma(L[13], tmp1, L[2]);
	L[2] = fma(L[23], tmp1 * dX[0] * float(5.000000e-01), L[2]);
	tmp1 = dX[1] * dX[1];
	L[2] = fma(L[16], tmp1 * float(5.000000e-01), L[2]);
	L[2] = fma(L[26], tmp1 * dX[0] * float(5.000000e-01), L[2]);
	L[2] = fma(L[30], tmp1 * dX[1] * float(1.666667e-01), L[2]);
	L[2] = fma(L[8], dX[2], L[2]);
	tmp1 = dX[2] * dX[0];
	L[2] = fma(L[14], tmp1, L[2]);
	L[2] = fma(L[24], tmp1 * dX[0] * float(5.000000e-01), L[2]);
	tmp1 = dX[2] * dX[1];
	L[2] = fma(L[17], tmp1, L[2]);
	L[2] = fma(L[27], tmp1 * dX[0], L[2]);
	L[2] = fma(L[31], tmp1 * dX[1] * float(5.000000e-01), L[2]);
	tmp1 = dX[2] * dX[2];
	L[2] = fma(L[18], tmp1 * float(5.000000e-01), L[2]);
	L[2] = fma(L[28], tmp1 * dX[0] * float(5.000000e-01), L[2]);
	L[2] = fma(L[32], tmp1 * dX[1] * float(5.000000e-01), L[2]);
	L[2] = fma(L[33], tmp1 * dX[2] * float(1.666667e-01), L[2]);
	L[3] = fma(L[6], dX[0], L[3]);
	tmp1 = dX[0] * dX[0];
	L[3] = fma(L[12], tmp1 * float(5.000000e-01), L[3]);
	L[3] = fma(L[22], tmp1 * dX[0] * float(1.666667e-01), L[3]);
	L[3] = fma(L[8], dX[1], L[3]);
	tmp1 = dX[1] * dX[0];
	L[3] = fma(L[14], tmp1, L[3]);
	L[3] = fma(L[24], tmp1 * dX[0] * float(5.000000e-01), L[3]);
	tmp1 = dX[1] * dX[1];
	L[3] = fma(L[17], tmp1 * float(5.000000e-01), L[3]);
	L[3] = fma(L[27], tmp1 * dX[0] * float(5.000000e-01), L[3]);
	L[3] = fma(L[31], tmp1 * dX[1] * float(1.666667e-01), L[3]);
	L[3] = fma(L[9], dX[2], L[3]);
	tmp1 = dX[2] * dX[0];
	L[3] = fma(L[15], tmp1, L[3]);
	L[3] = fma(L[25], tmp1 * dX[0] * float(5.000000e-01), L[3]);
	tmp1 = dX[2] * dX[1];
	L[3] = fma(L[18], tmp1, L[3]);
	L[3] = fma(L[28], tmp1 * dX[0], L[3]);
	L[3] = fma(L[32], tmp1 * dX[1] * float(5.000000e-01), L[3]);
	tmp1 = dX[2] * dX[2];
	L[3] = fma(L[19], tmp1 * float(5.000000e-01), L[3]);
	L[3] = fma(L[29], tmp1 * dX[0] * float(5.000000e-01), L[3]);
	L[3] = fma(L[33], tmp1 * dX[1] * float(5.000000e-01), L[3]);
	L[3] = fma(L[34], tmp1 * dX[2] * float(1.666667e-01), L[3]);
	L[4] = fma(L[10], dX[0], L[4]);
	L[4] = fma(L[20], dX[0] * dX[0] * float(5.000000e-01), L[4]);
	L[4] = fma(L[11], dX[1], L[4]);
	L[4] = fma(L[21], dX[1] * dX[0], L[4]);
	L[4] = fma(L[23], dX[1] * dX[1] * float(5.000000e-01), L[4]);
	L[4] = fma(L[12], dX[2], L[4]);
	L[4] = fma(L[22], dX[2] * dX[0], L[4]);
	L[4] = fma(L[24], dX[2] * dX[1], L[4]);
	L[4] = fma(L[25], dX[2] * dX[2] * float(5.000000e-01), L[4]);
	L[5] = fma(L[11], dX[0], L[5]);
	L[5] = fma(L[21], dX[0] * dX[0] * float(5.000000e-01), L[5]);
	L[5] = fma(L[13], dX[1], L[5]);
	L[5] = fma(L[23], dX[1] * dX[0], L[5]);
	L[5] = fma(L[26], dX[1] * dX[1] * float(5.000000e-01), L[5]);
	L[5] = fma(L[14], dX[2], L[5]);
	L[5] = fma(L[24], dX[2] * dX[0], L[5]);
	L[5] = fma(L[27], dX[2] * dX[1], L[5]);
	L[5] = fma(L[28], dX[2] * dX[2] * float(5.000000e-01), L[5]);
	L[7] = fma(L[13], dX[0], L[7]);
	L[7] = fma(L[23], dX[0] * dX[0] * float(5.000000e-01), L[7]);
	L[7] = fma(L[16], dX[1], L[7]);
	L[7] = fma(L[26], dX[1] * dX[0], L[7]);
	L[7] = fma(L[30], dX[1] * dX[1] * float(5.000000e-01), L[7]);
	L[7] = fma(L[17], dX[2], L[7]);
	L[7] = fma(L[27], dX[2] * dX[0], L[7]);
	L[7] = fma(L[31], dX[2] * dX[1], L[7]);
	L[7] = fma(L[32], dX[2] * dX[2] * float(5.000000e-01), L[7]);
	L[6] = fma(L[12], dX[0], L[6]);
	L[6] = fma(L[22], dX[0] * dX[0] * float(5.000000e-01), L[6]);
	L[6] = fma(L[14], dX[1], L[6]);
	L[6] = fma(L[24], dX[1] * dX[0], L[6]);
	L[6] = fma(L[27], dX[1] * dX[1] * float(5.000000e-01), L[6]);
	L[6] = fma(L[15], dX[2], L[6]);
	L[6] = fma(L[25], dX[2] * dX[0], L[6]);
	L[6] = fma(L[28], dX[2] * dX[1], L[6]);
	L[6] = fma(L[29], dX[2] * dX[2] * float(5.000000e-01), L[6]);
	L[8] = fma(L[14], dX[0], L[8]);
	L[8] = fma(L[24], dX[0] * dX[0] * float(5.000000e-01), L[8]);
	L[8] = fma(L[17], dX[1], L[8]);
	L[8] = fma(L[27], dX[1] * dX[0], L[8]);
	L[8] = fma(L[31], dX[1] * dX[1] * float(5.000000e-01), L[8]);
	L[8] = fma(L[18], dX[2], L[8]);
	L[8] = fma(L[28], dX[2] * dX[0], L[8]);
	L[8] = fma(L[32], dX[2] * dX[1], L[8]);
	L[8] = fma(L[33], dX[2] * dX[2] * float(5.000000e-01), L[8]);
	L[9] = fma(L[15], dX[0], L[9]);
	L[9] = fma(L[25], dX[0] * dX[0] * float(5.000000e-01), L[9]);
	L[9] = fma(L[18], dX[1], L[9]);
	L[9] = fma(L[28], dX[1] * dX[0], L[9]);
	L[9] = fma(L[32], dX[1] * dX[1] * float(5.000000e-01), L[9]);
	L[9] = fma(L[19], dX[2], L[9]);
	L[9] = fma(L[29], dX[2] * dX[0], L[9]);
	L[9] = fma(L[33], dX[2] * dX[1], L[9]);
	L[9] = fma(L[34], dX[2] * dX[2] * float(5.000000e-01), L[9]);
	L[10] = fma(L[20], dX[0], L[10]);
	L[10] = fma(L[21], dX[1], L[10]);
	L[10] = fma(L[22], dX[2], L[10]);
	L[11] = fma(L[21], dX[0], L[11]);
	L[11] = fma(L[23], dX[1], L[11]);
	L[11] = fma(L[24], dX[2], L[11]);
	L[13] = fma(L[23], dX[0], L[13]);
	L[13] = fma(L[26], dX[1], L[13]);
	L[13] = fma(L[27], dX[2], L[13]);
	L[16] = fma(L[26], dX[0], L[16]);
	L[16] = fma(L[30], dX[1], L[16]);
	L[16] = fma(L[31], dX[2], L[16]);
	L[12] = fma(L[22], dX[0], L[12]);
	L[12] = fma(L[24], dX[1], L[12]);
	L[12] = fma(L[25], dX[2], L[12]);
	L[14] = fma(L[24], dX[0], L[14]);
	L[14] = fma(L[27], dX[1], L[14]);
	L[14] = fma(L[28], dX[2], L[14]);
	L[17] = fma(L[27], dX[0], L[17]);
	L[17] = fma(L[31], dX[1], L[17]);
	L[17] = fma(L[32], dX[2], L[17]);
	L[15] = fma(L[25], dX[0], L[15]);
	L[15] = fma(L[28], dX[1], L[15]);
	L[15] = fma(L[29], dX[2], L[15]);
	L[18] = fma(L[28], dX[0], L[18]);
	L[18] = fma(L[32], dX[1], L[18]);
	L[18] = fma(L[33], dX[2], L[18]);
	L[19] = fma(L[29], dX[0], L[19]);
	L[19] = fma(L[33], dX[1], L[19]);
	L[19] = fma(L[34], dX[2], L[19]);

	/*#ifdef __CUDA_ARCH__
	 const auto& Lfactor = Lfactor_gpu;
	 #else
	 const auto& Lfactor = Lfactor_cpu;
	 #endif
	 for (int a = 0; a < 3; a++) {
	 me() += me(a) * dX[a];
	 for (int b = 0; b <= a; b++) {
	 me() += me(a, b) * dX[a] * dX[b] * Lfactor(a, b);
	 for (int c = 0; c <= b; c++) {
	 me() += me(a, b, c) * dX[a] * dX[b] * dX[c] * Lfactor(a, b, c);
	 for (int d = 0; d <= c; d++) {
	 me() += me(a, b, c, d) * dX[a] * dX[b] * dX[c] * dX[d] * Lfactor(a, b, c, d);
	 }
	 }
	 }
	 }
	 for (int a = 0; a < 3; a++) {
	 for (int b = 0; b < 3; b++) {
	 me(a) += me(a, b) * dX[b];
	 for (int c = 0; c <= b; c++) {
	 me(a) += me(a, b, c) * dX[b] * dX[c] * Lfactor(b, c);
	 for (int d = 0; d <= c; d++) {
	 me(a) += me(a, b, c, d) * dX[b] * dX[c] * dX[d] * Lfactor(b, c, d);
	 }
	 }
	 }
	 }
	 for (int a = 0; a < 3; a++) {
	 for (int b = 0; b <= a; b++) {
	 for (int c = 0; c < NDIM; c++) {
	 me(a, b) += me(a, b, c) * dX[c];
	 for (int d = 0; d <= c; d++) {
	 me(a, b) += me(a, b, c, d) * dX[c] * dX[d] * Lfactor(c, d);
	 }
	 }
	 }
	 }

	 for (int a = 0; a < 3; a++) {
	 for (int b = 0; b <= a; b++) {
	 for (int c = 0; c <= b; c++) {
	 for (int d = 0; d < 3; d++) {
	 me(a, b, c) += me(a, b, c, d) * dX[d];
	 }
	 }
	 }
	 }
	 */
	return L;
}

CUDA_EXPORT void shift_expansion(expansion<float> &me, array<float, NDIM> &g, float &phi,
		const array<float, NDIM> &dX) {
#ifdef __CUDA_ARCH__
	const auto& Lfactor = Lfactor_gpu;
#else
	const auto& Lfactor = Lfactor_cpu;
#endif
	phi = me();
	for (int a = 0; a < 3; a++) {
		phi += me(a) * dX[a];
		for (int b = a; b < 3; b++) {
			phi += me(a, b) * dX[a] * dX[b] * Lfactor(a, b);
			for (int c = b; c < 3; c++) {
				phi += me(a, b, c) * dX[a] * dX[b] * dX[c] * Lfactor(a, b, c);
				for (int d = c; d < 3; d++) {
					phi += me(a, b, c, d) * dX[a] * dX[b] * dX[c] * dX[d] * Lfactor(a, b, c, d);
				}
			}
		}
	}
	for (int a = 0; a < 3; a++) {
		g[a] = -me(a);
		for (int b = 0; b < 3; b++) {
			g[a] -= me(a, b) * dX[b];
			for (int c = b; c < 3; c++) {
				g[a] -= me(a, b, c) * dX[b] * dX[c] * Lfactor(b, c);
				for (int d = c; d < 3; d++) {
					g[a] -= me(a, b, c, d) * dX[b] * dX[c] * dX[d] * Lfactor(b, c, d);
				}
			}
		}
	}
}
